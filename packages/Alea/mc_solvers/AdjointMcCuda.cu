#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   AdjointMcCuda.cu
 * \author Steven Hamilton
 * \brief  Perform single history of adjoint MC
 */
//---------------------------------------------------------------------------//

#include <iterator>
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/generate.h>
#include <thrust/random.h>

#include "AdjointMcCuda.hh"
#include "utils/String_Functions.hh"
#include "harness/Warnings.hh"

#include "CudaUtils.hh"

namespace alea
{

#ifndef USE_LDG
#define USE_LDG 0
#endif

//---------------------------------------------------------------------------//
/*!
 * \brief Initialize history into new state
 */
//---------------------------------------------------------------------------//
__device__ void initializeHistory(int &state, double &wt, int N,
        const double * const start_cdf,
        const double * const start_wt,
              hiprandState   *rng_state)
{
    // Generate random number
    double rand = hiprand_uniform_double(rng_state);

    // Sample cdf to get new state
    auto elem = lower_bound(start_cdf,start_cdf+N,rand);

    if( elem == &start_cdf[N-1]+1 )
    {
        state = -1;
        wt    = 0.0;
        return;
    }

    // Get weight and update state
    state = elem-start_cdf;
#if USE_LDG
    wt    = __ldg(&start_wt[state]); //modified by Max
#else
    wt = start_wt[state];
#endif
}


__device__ void initializeHistory2(int &state, double &wt, int N,
        const double * const start_cdf,
        const double * const start_wt,
              double &rand)
{

    // Sample cdf to get new state
    auto elem = lower_bound(start_cdf,start_cdf+N,rand);

    if( elem == &start_cdf[N-1]+1 )
    {
        state = -1;
        wt    = 0.0;
        return;
    }

    // Get weight and update state
    state = elem-start_cdf;
#if USE_LDG
    wt    = __ldg(&start_wt[state]); //modified by Max
#else
    wt = start_wt[state];
#endif
}



//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//
__device__ void tallyContribution(int state, double wt,
              double * const x, 
        const double * const H,
        const int    * const inds,
        const int    * const offsets,
              bool           expected_value)
{
    if( expected_value )
    {
        int row_begin = offsets[state];
        int row_end   = offsets[state+1];

        // For expected value estimator, loop over current row and add
        // contributions corresponding to each element
        for( int i=row_begin; i<row_end; ++i )
        {
#if USE_LDG
            atomicAdd(x+inds[i],wt* ( __ldg(&H[i]) ) );//modified by Max
#else
            atomicAdd(x+inds[i],wt*H[i]);//modified by Max
#endif

        }
    }
    else
    {
        // Collision estimator just adds weight
        atomicAdd(x+state,wt);
    }
}


//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//
__global__ void adjoint_run_monte_carlo(int N, int history_length, double wt_cutoff,
        int batch_size,
        bool expected_value,
        const double * const start_cdf,
        const double * const start_wt,
        const double * const H,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
        const double * const coeffs,
              double * const x,
              hiprandState   *rng_state)
{
    int state = -1;
    double wt = 0.0;

    // Store rng state locally
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = rng_state[tid];
 
    extern __shared__ double steps[];
 
    for (int i = 0; i<batch_size; ++i)
        steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);



    // Get initial state for this history by sampling from start_cdf
    initializeHistory(state,wt,N,start_cdf,start_wt,&local_state);

    //initializeHistory2(state,wt,N,start_cdf,start_wt,steps[threadIdx.x]);

    //printf("Starting history in state %i with weight %7.3f\n",state,wt);
    if( state == -1 )
    {
        rng_state[tid] = local_state;
        return;
    }
    double init_wt = wt;

    // With expected value estimator we start on stage 1 because
    // zeroth order term is added explicitly at the end
    int stage = expected_value ? 1 : 0;

    // Perform initial tally
    tallyContribution(state,coeffs[stage]*wt,x,H,inds,offsets,
        expected_value);

    int count_batch = 0;

    for( ; stage<=history_length; ++stage )
    {
        if (count_batch == batch_size)
        {

          //__syncthreads();
         count_batch = 0;
         for (int i = 0; i<batch_size; ++i)
            steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);
        }

        // Move to new state
        //getNewState(state,wt,P,W,inds,offsets,&local_state);
        //printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

        getNewState2(state,wt,P,W,inds,offsets,steps[threadIdx.x + count_batch * blockDim.x]);

        if( state == -1 )
            break;

        // Tally
        tallyContribution(state,coeffs[stage]*wt,x,H,inds,offsets,
            expected_value);

        count_batch++;

        // Check weight cutoff
        if( std::abs(wt/init_wt) < wt_cutoff )
            break;
   
    }

    // Store rng state back to global
    rng_state[tid] = local_state;
}
        
//---------------------------------------------------------------------------//
/*!
 * \brief Constructor
 *
 * \param P Views into entries of probability matrix
 * \param W Views into entries of weight matrix
 * \param inds Views into nonzeros indices
 * \param offsets Starting indices for each matrix row
 * \param coeffs Polynomial coefficients
 * \param pl Problem parameters
 */
//---------------------------------------------------------------------------//
AdjointMcCuda::AdjointMcCuda(
        Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view     coeffs,
        Teuchos::RCP<Teuchos::ParameterList> pl)

  : d_N(mc_data->getIterationMatrix()->getGlobalNumRows())
{
    // Get parameters
    d_num_histories      = pl->get("num_histories",1000);
    d_max_history_length = coeffs.size()-1;
    d_weight_cutoff      = pl->get("weight_cutoff",0.0);

    // Determine type of tally
    std::string estimator = pl->get<std::string>("estimator",
                                                 "expected_value");
    VALIDATE(estimator == "collision" ||
             estimator == "expected_value",
             "Only collision and expected_value estimators are available.");
    d_use_expected_value = (estimator == "expected_value");

    // Should we print anything to screen
    std::string verb = profugus::to_lower(pl->get("verbosity","low"));
    if( verb == "none" )
        d_verbosity = NONE;
    else if( verb == "low" )
        d_verbosity = LOW;
    else if( verb == "high" )
        d_verbosity = HIGH;

    prepareDeviceData(mc_data,coeffs);

    d_num_curand_calls = 0;
    d_rng_seed = pl->get<int>("rng_seed",1234);
}

//---------------------------------------------------------------------------//
// Solve problem using Monte Carlo
//---------------------------------------------------------------------------//
void AdjointMcCuda::solve(const MV &b, MV &x)
{
    // Containers to hold starting cdf and wt arrays
    thrust::device_vector<double> start_cdf, start_wt;

    // Build initial probability and weight distributions
    Teuchos::ArrayRCP<const double> b_data = b.getData(0);
    build_initial_distribution(b_data,start_cdf,start_wt);

    // Get pointers for kernel
    const double * const start_cdf_ptr =
        thrust::raw_pointer_cast(start_cdf.data());
    const double * const start_wt_ptr  =
        thrust::raw_pointer_cast(start_wt.data());

    const double * const H       = thrust::raw_pointer_cast(d_H.data());
    const double * const P       = thrust::raw_pointer_cast(d_P.data());
    const double * const W       = thrust::raw_pointer_cast(d_W.data());
    const int    * const inds    = thrust::raw_pointer_cast(d_inds.data());
    const int    * const offsets = thrust::raw_pointer_cast(d_offsets.data());
    const double * const coeffs  = thrust::raw_pointer_cast(d_coeffs.data());

    // Create vector for state
    thrust::device_vector<double> x_vec(d_N);
    double * const x_ptr = thrust::raw_pointer_cast(x_vec.data());

    int block_size = std::min(256,d_num_histories);
    int num_blocks = d_num_histories / block_size;

    hiprandState *rng_states;
    hipError_t e = hipMalloc((void **)&rng_states,
        block_size*num_blocks*sizeof(hiprandState));

    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to allocate memory");

    // Initialize RNG
    //initialize_rng<<<num_blocks,block_size>>>(rng_states,d_rng_seed,
    //    d_num_curand_calls);

    thrust::device_vector<int> seeds( block_size*num_blocks);
    thrust::sequence(seeds.begin(), seeds.end(), d_rng_seed);
    int* seed_ptr = thrust::raw_pointer_cast(seeds.data());

    initialize_rng2<<<num_blocks, block_size>>>(rng_states, seed_ptr, 
          d_num_curand_calls);

    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to initialize RNG");
    d_num_curand_calls++;

    int batch_size = 5;

    adjoint_run_monte_carlo<<< num_blocks,block_size, sizeof(double) * block_size * batch_size >>>(d_N,d_max_history_length,
        d_weight_cutoff, batch_size, d_use_expected_value,
        start_cdf_ptr,start_wt_ptr,H,P,W,inds,offsets,coeffs,x_ptr,rng_states);

    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to execute MC kernel");

    // Scale by history count
    for( auto itr= x_vec.begin(); itr != x_vec.end(); ++itr )
        *itr /= static_cast<double>(num_blocks*block_size);

    // Copy data back to host
    {
        Teuchos::ArrayRCP<double> x_data = x.getDataNonConst(0);
        thrust::copy(x_vec.begin(),x_vec.end(),x_data.get());
    }


    // Add rhs for expected value
    if( d_use_expected_value )
       x.update(d_coeffs[0],b,1.0);

    // Free RNG state
    e = hipFree(rng_states);
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to deallocate memory");
}

//---------------------------------------------------------------------------//
// PRIVATE FUNCTIONS
//---------------------------------------------------------------------------//

//---------------------------------------------------------------------------//
// Extract matrices into ArrayView objects for faster data access
//---------------------------------------------------------------------------//
void AdjointMcCuda::prepareDeviceData(Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view coeffs)
{
    Teuchos::RCP<const MATRIX> H = mc_data->getIterationMatrix();
    Teuchos::RCP<const MATRIX> P = mc_data->getProbabilityMatrix();
    Teuchos::RCP<const MATRIX> W = mc_data->getWeightMatrix();

    d_nnz = H->getNodeNumEntries();
    d_H.resize(d_nnz);
    d_P.resize(d_nnz);
    d_W.resize(d_nnz);
    d_inds.resize(d_nnz);
    d_offsets.resize(d_N+1);

    Teuchos::ArrayView<const double> val_row;
    Teuchos::ArrayView<const int>    ind_row;
    auto h_iter   = d_H.begin();
    auto p_iter   = d_P.begin();
    auto w_iter   = d_W.begin();
    auto ind_iter = d_inds.begin();
    // This loop should perhaps be rewritten, right now a separate call
    // to hipMemcpy is performed for each row of each matrix
    // It might be more efficient to create a single vector on the CPU
    // and do a single copy to device?
    d_offsets[0] = 0;
    for( int i=0; i<d_N; ++i )
    {
        // Extract row i of matrix
        H->getLocalRowView(i,ind_row,val_row);
        thrust::copy(val_row.begin(),val_row.end(),h_iter);
        h_iter += val_row.size();
        P->getLocalRowView(i,ind_row,val_row);
        thrust::copy(val_row.begin(),val_row.end(),p_iter);
        p_iter += val_row.size();
        W->getLocalRowView(i,ind_row,val_row);
        thrust::copy(val_row.begin(),val_row.end(),w_iter);
        w_iter += val_row.size();
        thrust::copy(ind_row.begin(),ind_row.end(),ind_iter);
        ind_iter += ind_row.size();
        d_offsets[i+1] = d_offsets[i] + ind_row.size();
    }
    CHECK( h_iter   == d_H.end() );
    CHECK( p_iter   == d_P.end() );
    CHECK( w_iter   == d_W.end() );
    CHECK( ind_iter == d_inds.end() );

    // Copy coefficients into device vector
    const_scalar_view::HostMirror coeffs_host = Kokkos::create_mirror_view(coeffs);
    Kokkos::deep_copy(coeffs_host,coeffs);
    d_coeffs.resize(coeffs.size());
    thrust::copy(coeffs_host.ptr_on_device(),
                 coeffs_host.ptr_on_device()+coeffs_host.size(),
                 d_coeffs.begin());
}

//---------------------------------------------------------------------------//
// Build initial cdf and weights
//---------------------------------------------------------------------------//
void AdjointMcCuda::build_initial_distribution(
        Teuchos::ArrayRCP<const double> b,
        thrust::device_vector<double>  &cdf,
        thrust::device_vector<double>  &wt) const
{
    thrust::host_vector<double> cdf_host(d_N);
    thrust::host_vector<double> wt_host(d_N);

    // First take absolute value of b
    for( int i=0; i<d_N; ++i )
    {
        cdf_host[i] = std::abs(b[i]);
    }

    // Normalize to get a PDF
    double pdf_sum = std::accumulate(cdf_host.begin(),cdf_host.end(),0.0);
    ENSURE( pdf_sum > 0.0 );
    std::transform(cdf_host.begin(),cdf_host.end(),cdf_host.begin(),
            [pdf_sum](double val){return val/pdf_sum;});

    // Compute weight vector s.t. b = p * wt_host
    std::transform(b.begin(),b.end(),cdf_host.begin(),wt_host.begin(),
            [](double u, double v){return v==0.0 ? 0.0 : u/v;});

    // Convert PDF to CDF
    std::partial_sum(cdf_host.begin(),cdf_host.end(),cdf_host.begin());

    // Copy data to device
    cdf = cdf_host;
    wt  = wt_host;
}

} // namespace alea
