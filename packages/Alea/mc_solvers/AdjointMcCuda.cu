#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   AdjointMcCuda.cu
 * \author Steven Hamilton
 * \brief  Perform single history of adjoint MC
 */
//---------------------------------------------------------------------------//

#include <iterator>
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/generate.h>
#include <thrust/random.h>

#include "AdjointMcCuda.hh"
#include "utils/String_Functions.hh"
#include "harness/Warnings.hh"

namespace alea
{

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif 

#ifndef BATCH_SIZE
#define BATCH_SIZE 5
#endif

//---------------------------------------------------------------------------//
/*!
 * \brief Initialize history into new state
 */
//---------------------------------------------------------------------------//
__device__ void initializeHistory(int &state, double &wt, int N,
        const double * const start_cdf,
        const double * const start_wt,
              hiprandState   *rng_state)
{
    // Generate random number
    double rand = hiprand_uniform_double(rng_state);

    // Sample cdf to get new state
    auto elem = lower_bound(start_cdf,start_cdf+N,rand);

    if( elem == &start_cdf[N-1]+1 )
    {
        state = -1;
        wt    = 0.0;
        return;
    }

    // Get weight and update state
    state = elem-start_cdf;
#if USE_LDG
    wt    = __ldg(&start_wt[state]); //modified by Max
#else
    wt = start_wt[state];
#endif
}


__device__ void initializeHistory2(int &state, double &wt, int N,
        const double * const start_cdf,
        const double * const start_wt,
              double &rand)
{

    // Sample cdf to get new state
    auto elem = lower_bound(start_cdf,start_cdf+N,rand);

    if( elem == &start_cdf[N-1]+1 )
    {
        state = -1;
        wt    = 0.0;
        return;
    }

    // Get weight and update state
    state = elem-start_cdf;
#if USE_LDG
    wt    = __ldg(&start_wt[state]); //modified by Max
#else
    wt = start_wt[state];
#endif
}



//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//
__device__ void tallyContribution(int state, double wt,
              double * const x, 
        const double * const H,
        const int    * const inds,
        const int    * const offsets,
              bool           expected_value)
{
    if( expected_value )
    {
        int row_begin = offsets[state];
        int row_end   = offsets[state+1];

        // For expected value estimator, loop over current row and add
        // contributions corresponding to each element
        for( int i=row_begin; i<row_end; ++i )
        {
#if USE_LDG
            atomicAdd(x+inds[i],wt* ( __ldg(&H[i]) ) );//modified by Max
#else
            atomicAdd(x+inds[i],wt*H[i]);//modified by Max
#endif

        }
    }
    else
    {
        // Collision estimator just adds weight
        atomicAdd(x+state,wt);
    }
}


__device__ void tallyContribution(int state, double wt,
              double * const x, 
              const device_row_data * data,
              const int    * const offsets,
              bool           expected_value)
{
    if( expected_value )
    {
        int row_begin = offsets[state];
        int row_end   = offsets[state+1];

        // For expected value estimator, loop over current row and add
        // contributions corresponding to each element
        for( int i=row_begin; i<row_end; ++i )
        {
#if USE_LDG
            atomicAdd(x+data[i].inds,
             wt* ( __ldg(&(data[i].H)) ));//modified by Max
#else
            atomicAdd(x+data[i].inds,
                wt*data[i].H);//modified by Max
#endif

        }
    }
    else
    {
        // Collision estimator just adds weight
        atomicAdd(x+state,wt);
    }
}

__global__ void run_adjoint_monte_carlo(int N, int history_length, double wt_cutoff,
        bool expected_value,
        const double * const start_cdf,
        const double * const start_wt,
        const double * const H,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
        const double * const coeffs,
              double * const x,
              hiprandState   *rng_state)
{
    int state = -1;
    double wt = 0.0;

    // Store rng state locally
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = rng_state[tid];
 
/*    __shared__ double steps[BLOCK_SIZE * BATCH_SIZE];
 
    for (int i = 0; i<BATCH_SIZE; ++i)
        steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);
*/
    // Get initial state for this history by sampling from start_cdf
    initializeHistory(state,wt,N,start_cdf,start_wt,&local_state);

    //initializeHistory2(state,wt,N,start_cdf,start_wt,steps[threadIdx.x]);

    //printf("Starting history in state %i with weight %7.3f\n",state,wt);
    if( state == -1 )
    {
        rng_state[tid] = local_state;
        return;
    }
    double init_wt = wt;

    // With expected value estimator we start on stage 1 because
    // zeroth order term is added explicitly at the end
    int stage = expected_value ? 1 : 0;

    // Perform initial tally
    tallyContribution(state,coeffs[stage]*wt,x,H,inds,offsets,
        expected_value);

    //int count_batch = 1;

    for( ; stage<=history_length; ++stage )
    {
    /*    if (count_batch == BATCH_SIZE)
        {

          //__syncthreads();
         count_batch = 0;
         for (int i = 0; i<BATCH_SIZE; ++i)
            steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);
        }*/

        // Move to new state
        getNewState(state,wt,P,W,inds,offsets,&local_state);
        //printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

        //getNewState2(state,wt,P,W,inds,offsets,steps[threadIdx.x + count_batch * blockDim.x]);

        if( state == -1 )
            break;

        // Tally
        tallyContribution(state,coeffs[stage]*wt,x,H,inds,offsets,
            expected_value);

        //count_batch++;

        // Check weight cutoff
        if( std::abs(wt/init_wt) < wt_cutoff )
            break;
   
    }

    // Store rng state back to global
    rng_state[tid] = local_state;
}

//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//
__global__ void run_adjoint_monte_carlo(int N, int history_length, double wt_cutoff,
        bool expected_value,
        const double * const start_cdf,
        const double * const start_wt,
        device_row_data * data,
        const int    * const offsets,
        const double * const coeffs,
              double * const x,
              hiprandState   *rng_state)
{
    int state = -1;
    double wt = 0.0;

    // Store rng state locally
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = rng_state[tid];
 
    // Get initial state for this history by sampling from start_cdf
    initializeHistory(state,wt,N,start_cdf,start_wt,&local_state);

    //printf("Starting history in state %i with weight %7.3f\n",state,wt);
    if( state == -1 )
    {
        rng_state[tid] = local_state;
        return;
    }
    double init_wt = wt;

    // With expected value estimator we start on stage 1 because
    // zeroth order term is added explicitly at the end
    int stage = expected_value ? 1 : 0;

    // Perform initial tally
    tallyContribution(state,coeffs[stage]*wt,x,data,offsets,
        expected_value);

    for( ; stage<=history_length; ++stage )
    {

        // Move to new state
        getNewState(state,wt,data,offsets,&local_state);
        //printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

        if( state == -1 )
            break;

        // Tally
        tallyContribution(state,coeffs[stage]*wt,x,data,offsets,
            expected_value);

        // Check weight cutoff
        if( std::abs(wt/init_wt) < wt_cutoff )
            break;
   
    }

    // Store rng state back to global
    rng_state[tid] = local_state;
}



        
//---------------------------------------------------------------------------//
/*!
 * \brief Constructor
 *
 * \param P Views into entries of probability matrix
 * \param W Views into entries of weight matrix
 * \param inds Views into nonzeros indices
 * \param offsets Starting indices for each matrix row
 * \param coeffs Polynomial coefficients
 * \param pl Problem parameters
 */
//---------------------------------------------------------------------------//
AdjointMcCuda::AdjointMcCuda(
        Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view     coeffs,
        Teuchos::RCP<Teuchos::ParameterList> pl)

  : d_N(mc_data->getIterationMatrix()->getGlobalNumRows())
{
    // Get parameters
    d_num_histories      = pl->get("num_histories",1000);
    d_max_history_length = coeffs.size()-1;
    d_weight_cutoff      = pl->get("weight_cutoff",0.0);
    d_struct             = pl->get("struct_matrix", 0);

    VALIDATE( d_struct==0 || d_struct==1, 
            "Value for the flag to manage matrix data not valid" );

    // Determine type of tally
    std::string estimator = pl->get<std::string>("estimator",
                                                 "expected_value");
    VALIDATE(estimator == "collision" ||
             estimator == "expected_value",
             "Only collision and expected_value estimators are available.");
    d_use_expected_value = (estimator == "expected_value");

    // Should we print anything to screen
    std::string verb = profugus::to_lower(pl->get("verbosity","low"));
    if( verb == "none" )
        d_verbosity = NONE;
    else if( verb == "low" )
        d_verbosity = LOW;
    else if( verb == "high" )
        d_verbosity = HIGH;

    prepareDeviceData(mc_data,coeffs);

    d_num_curand_calls = 0;
    d_rng_seed = pl->get<int>("rng_seed",1234);
}

//---------------------------------------------------------------------------//
// Solve problem using Monte Carlo
//---------------------------------------------------------------------------//
void AdjointMcCuda::solve(const MV &b, MV &x)
{
    // Containers to hold starting cdf and wt arrays
    thrust::device_vector<double> start_cdf, start_wt;

    // Build initial probability and weight distributions
    Teuchos::ArrayRCP<const double> b_data = b.getData(0);
    build_initial_distribution(b_data,start_cdf,start_wt);

    // Get pointers for kernel
    const double * const start_cdf_ptr =
        thrust::raw_pointer_cast(start_cdf.data());
    const double * const start_wt_ptr  =
        thrust::raw_pointer_cast(start_wt.data());

    const double * H;
    const double * P;
    const double * W;
    const int    * inds; 
    device_row_data * data_ptr;

    if( d_struct==0 )
    {
    	H       = thrust::raw_pointer_cast(d_H.data());
    	P       = thrust::raw_pointer_cast(d_P.data());
    	W       = thrust::raw_pointer_cast(d_W.data());
    	inds    = thrust::raw_pointer_cast(d_inds.data());
    }
    else
    	 data_ptr = thrust::raw_pointer_cast(mat_data.data());   

    const int    * const offsets = thrust::raw_pointer_cast(d_offsets.data());
    const double * const coeffs  = thrust::raw_pointer_cast(d_coeffs.data());

    // Create vector for state
    thrust::device_vector<double> x_vec(d_N);
    double * const x_ptr = thrust::raw_pointer_cast(x_vec.data());

    //int block_size = std::min(256,d_num_histories);
    VALIDATE( BLOCK_SIZE <= d_num_histories, 
          "Number of histories is smaller than the block size" );

    int num_blocks = d_num_histories / BLOCK_SIZE;

    hiprandState *rng_states;
    hipError_t e = hipMalloc( (void **)&rng_states,
        BLOCK_SIZE * num_blocks * sizeof(hiprandState) );

    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to allocate memory");

    // Initialize RNG
    //initialize_rng<<<num_blocks,block_size>>>(rng_states,d_rng_seed,
    //    d_num_curand_calls);

    thrust::device_vector<int> seeds( BLOCK_SIZE * num_blocks );
    thrust::sequence(seeds.begin(), seeds.end(), d_rng_seed);
    int* seed_ptr = thrust::raw_pointer_cast(seeds.data());

    initialize_rng2<<<num_blocks, BLOCK_SIZE>>>(rng_states, seed_ptr, 
          d_num_curand_calls);

    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to initialize RNG");
    d_num_curand_calls++;

    if( d_struct==0 )
    {
    	run_adjoint_monte_carlo<<< num_blocks,BLOCK_SIZE >>>(d_N,
                d_max_history_length, d_weight_cutoff, d_use_expected_value,
                start_cdf_ptr,start_wt_ptr,H,P,W,
                inds,offsets,coeffs,x_ptr,rng_states);
    }
    else
    {
        //std::cout<<"Qui ci arrivo"<<std::endl;
    	run_adjoint_monte_carlo<<< num_blocks,BLOCK_SIZE >>>(d_N,
                d_max_history_length, d_weight_cutoff, d_use_expected_value,
                start_cdf_ptr,start_wt_ptr,data_ptr,
                offsets,coeffs,x_ptr,rng_states);
    }

    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to execute MC kernel");

    // Scale by history count
    for( auto itr= x_vec.begin(); itr != x_vec.end(); ++itr )
        *itr /= static_cast<double>( num_blocks * BLOCK_SIZE );

    // Copy data back to host
    {
        Teuchos::ArrayRCP<double> x_data = x.getDataNonConst(0);
        thrust::copy(x_vec.begin(),x_vec.end(),x_data.get());
    }


    // Add rhs for expected value
    if( d_use_expected_value )
       x.update(d_coeffs[0],b,1.0);

    // Free RNG state
    e = hipFree(rng_states);
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to deallocate memory");
}

//---------------------------------------------------------------------------//
// PRIVATE FUNCTIONS
//---------------------------------------------------------------------------//

//---------------------------------------------------------------------------//
// Extract matrices into ArrayView objects for faster data access
//---------------------------------------------------------------------------//
void AdjointMcCuda::prepareDeviceData(Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view coeffs)
{
        Teuchos::RCP<const MATRIX> H = mc_data->getIterationMatrix();
    	Teuchos::RCP<const MATRIX> P = mc_data->getProbabilityMatrix();
    	Teuchos::RCP<const MATRIX> W = mc_data->getWeightMatrix();
        d_offsets.resize(d_N+1);

	if(d_struct == 0)
	{
   		d_nnz = H->getNodeNumEntries();
    		d_H.resize(d_nnz);
    		d_P.resize(d_nnz);
    		d_W.resize(d_nnz);
    		d_inds.resize(d_nnz);
	
    		Teuchos::ArrayView<const double> val_row;
    		Teuchos::ArrayView<const int>    ind_row;
    		auto h_iter   = d_H.begin();
    		auto p_iter   = d_P.begin();
    		auto w_iter   = d_W.begin();
    		auto ind_iter = d_inds.begin();
    // This loop should perhaps be rewritten, right now a separate call
    // to hipMemcpy is performed for each row of each matrix
    // It might be more efficient to create a single vector on the CPU
    // and do a single copy to device?
    		d_offsets[0] = 0;
    		for( int i=0; i<d_N; ++i )
    		{
        		// Extract row i of matrix
        		H->getLocalRowView(i,ind_row,val_row);
        		thrust::copy(val_row.begin(),val_row.end(),h_iter);
        		h_iter += val_row.size();
        		P->getLocalRowView(i,ind_row,val_row);
        		thrust::copy(val_row.begin(),val_row.end(),p_iter);
        		p_iter += val_row.size();
       			W->getLocalRowView(i,ind_row,val_row);
        		thrust::copy(val_row.begin(),val_row.end(),w_iter);
        		w_iter += val_row.size();
        		thrust::copy(ind_row.begin(),ind_row.end(),ind_iter);
        		ind_iter += ind_row.size();
        		d_offsets[i+1] = d_offsets[i] + ind_row.size();
    		}
    		CHECK( h_iter   == d_H.end() );
    		CHECK( p_iter   == d_P.end() );
    		CHECK( w_iter   == d_W.end() );
    		CHECK( ind_iter == d_inds.end() );

    		// Copy coefficients into device vector
    		const_scalar_view::HostMirror coeffs_host = Kokkos::create_mirror_view(coeffs);
    		Kokkos::deep_copy(coeffs_host,coeffs);
    		d_coeffs.resize(coeffs.size());
    		thrust::copy(coeffs_host.ptr_on_device(),
                	 coeffs_host.ptr_on_device()+coeffs_host.size(),
                 	d_coeffs.begin());
	}

	else
	{
            d_nnz = H->getNodeNumEntries();
    		Teuchos::ArrayView<const double> pval_row;
    		Teuchos::ArrayView<const double> hval_row;
    		Teuchos::ArrayView<const double> wval_row;
    		Teuchos::ArrayView<const int>    ind_row;
    // This loop should perhaps be rewritten, right now a separate call
    // to hipMemcpy is performed for each row of each matrix
    // It might be more efficient to create a single vector on the CPU
    // and do a single copy to device?
    		d_offsets[0] = 0;

    		thrust::host_vector< device_row_data > data_host( d_nnz );
    		mat_data.resize( d_nnz );

    		int count = 0;
    		for( int i=0; i<d_N; ++i )
    		{
        		// Extract row i of matrix
        		H->getLocalRowView(i,ind_row,hval_row);
        		P->getLocalRowView(i,ind_row,pval_row);
        		W->getLocalRowView(i,ind_row,wval_row); 
     
        		for( int j = 0; j < ind_row.size(); ++j )
        		{
                    CHECK( count < data_host.size() );
                    CHECK( j < hval_row.size() );
                    CHECK( j < pval_row.size() );
                    CHECK( j < wval_row.size() );
                    CHECK( j < ind_row.size() );
                    data_host[count].H   = hval_row[j];            
                    data_host[count].P   = pval_row[j];
                    data_host[count].W   = wval_row[j];
                    data_host[count].inds = ind_row[j];
                    count++;
        		}
        
        		d_offsets[i+1] = d_offsets[i] + ind_row.size();
    		}

    		CHECK( count == d_nnz );

    		thrust::copy( data_host.begin(), data_host.end(), mat_data.begin() );
            //mat_data = data_host;

    		// Copy coefficients into device vector
    		const_scalar_view::HostMirror coeffs_host = Kokkos::create_mirror_view(coeffs);
    		Kokkos::deep_copy(coeffs_host,coeffs);
    		d_coeffs.resize(coeffs.size());
    		thrust::copy(coeffs_host.ptr_on_device(),
                 	coeffs_host.ptr_on_device()+coeffs_host.size(),
                 	d_coeffs.begin());

       }
}
//---------------------------------------------------------------------------//
// Build initial cdf and weights
//---------------------------------------------------------------------------//
void AdjointMcCuda::build_initial_distribution(
        Teuchos::ArrayRCP<const double> b,
        thrust::device_vector<double>  &cdf,
        thrust::device_vector<double>  &wt) const
{
    thrust::host_vector<double> cdf_host(d_N);
    thrust::host_vector<double> wt_host(d_N);

    // First take absolute value of b
    for( int i=0; i<d_N; ++i )
    {
        cdf_host[i] = std::abs(b[i]);
    }

    // Normalize to get a PDF
    double pdf_sum = std::accumulate(cdf_host.begin(),cdf_host.end(),0.0);
    ENSURE( pdf_sum > 0.0 );
    std::transform(cdf_host.begin(),cdf_host.end(),cdf_host.begin(),
            [pdf_sum](double val){return val/pdf_sum;});

    // Compute weight vector s.t. b = p * wt_host
    std::transform(b.begin(),b.end(),cdf_host.begin(),wt_host.begin(),
            [](double u, double v){return v==0.0 ? 0.0 : u/v;});

    // Convert PDF to CDF
    std::partial_sum(cdf_host.begin(),cdf_host.end(),cdf_host.begin());

    // Copy data to device
    cdf = cdf_host;
    wt  = wt_host;
}

} // namespace alea
