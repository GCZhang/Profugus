#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   CudaUtils.cu
 * \author Steven Hamilton
 * \brief  Perform single history of adjoint MC
 */
//---------------------------------------------------------------------------//


#include "CudaUtils.hh"

#include <iterator>
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/generate.h>
#include <thrust/random.h>
#include "utils/String_Functions.hh"
#include "harness/Warnings.hh"


namespace alea
{


#ifndef USE_LDG
#define USE_LDG 0
#endif


// lower_bound implementation that can be called from device
__device__ const double * lower_bound(const double * first,
        const double * last,
        double   val)
{
    const double * it;
    int count, step;
    count = last - first;
    while( count > 0 )
    {
        step = count / 2;
        it = first+step;
#if USE_LDG
        if( __ldg( &(*it) ) < val ) //Modified by Max
#else
        if ( *it<val )
#endif
        {
            first = ++it;
            count -= step+1;
        }
        else
        {
            count = step;
        }
    }
    return first;
}

// atomicAdd, not provided by Cuda for doubles
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                    __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


//---------------------------------------------------------------------------//
/*!
 * \brief Get new state by sampling from cdf
 */
//---------------------------------------------------------------------------//
__device__ void getNewState(int &state, double &wt,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
              hiprandState   *rng_state )
{
    // Generate random number
    double rand = hiprand_uniform_double(rng_state);

    // Sample cdf to get new state
    auto beg_row = P + offsets[state];
    auto end_row = P + offsets[state+1];
    auto elem = lower_bound(beg_row,end_row,rand);
    //auto elem = thrust::lower_bound( thrust::seq, beg_row, end_row, rand);

    if( elem == end_row )
    {
        // Invalidate all row data
        state = -1;
        wt = 0.0;
        return;
    }

    // Modify weight and update state
    auto index = elem - P;
#if USE_LDG
    state  =  __ldg(&inds[index]); //modified by Max
    wt    *=  __ldg(&W[index]); //modified by Max
#else
    state = inds[index];
    wt *= W[index];
#endif
}


__device__ void getNewState2(int &state, double &wt,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
              double   &rand )
{

    // Sample cdf to get new state
    auto beg_row = P + offsets[state];
    auto end_row = P + offsets[state+1];
    auto elem = lower_bound(beg_row,end_row,rand);
    //auto elem = thrust::lower_bound( thrust::seq, beg_row, end_row, rand);

    if( elem == end_row )
    {
        // Invalidate all row data
        state = -1;
        wt = 0.0;
        return;
    }

    // Modify weight and update state
    auto index = elem - P;
#if USE_LDG
    state  =  __ldg(&inds[index]); //modified by Max
    wt    *=  __ldg(&W[index]); //modified by Max
#else
    state = inds[index];
    wt *= W[index];
#endif
}


//---------------------------------------------------------------------------//
/*!
 * \brief Initialize Cuda RNG
 */
//---------------------------------------------------------------------------//
__global__ void initialize_rng(hiprandState *state, int seed, int offset)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(seed,tid,offset,&state[tid]);

}


__global__ void initialize_rng2(hiprandState *state, int*seed, int offset)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(seed[tid], 0, offset, &state[tid]);
}



}
