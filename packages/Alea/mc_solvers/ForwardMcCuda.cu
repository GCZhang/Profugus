#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   ForwardMcCuda.cu
 * \author Steven Hamilton
 * \brief  Perform single history of adjoint MC
 */
//---------------------------------------------------------------------------//

#include <iterator>
#include <cmath>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/generate.h>
#include <thrust/random.h>

#include "ForwardMcCuda.hh"
#include "utils/String_Functions.hh"
#include "harness/Warnings.hh"


namespace alea
{

//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//
__device__ void tallyContribution(int state, double wt, double * const x)
{
        // Collision estimator just adds weight
        atomicAdd(x+state,wt);
}

__device__ void tallyContribution2(double wt, double * const x)
{
        atomicAdd(x,wt);
}


//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//

template<class MemoryAccess>
__global__ void run_forward_monte_carlo(int N, int history_length, double wt_cutoff,
        int entry_histories, 
        int batch_size,
        const double * const H,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
        const double * const coeffs,
              double * const x,
        const double * const rhs, 
              hiprandState   *rng_state)
{
    int state = -1;
    double wt = 1.0;

    // Store rng state locally
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int entry = tid / entry_histories;

    if(entry >= N)
      return;

    state = entry;
    hiprandState local_state = rng_state[tid];
 
/*    extern __shared__ double steps[];
 
    for (int i = 0; i<batch_size; ++i)
        steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);

*/


    //initializeHistory2(state,wt,N,start_cdf,start_wt,steps[threadIdx.x]);

    //printf("Starting history in state %i with weight %7.3f\n",state,wt);
    if( state >= N )
    {
        rng_state[tid] = local_state;
        return;
    }
    double init_wt = wt;

    int stage = 0;

    // Perform initial tally
    tallyContribution(state,coeffs[stage]*wt*rhs[state],x);

  //  int count_batch = 0;

    for(; stage<=history_length; ++stage )
    {
/*        if (count_batch == batch_size)
        {

          //__syncthreads();
         count_batch = 0;
         for (int i = 0; i<batch_size; ++i)
            steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);
        }

*/

        // Move to new state
        getNewState<MemoryAccess>(state,wt,P,W,inds,offsets,&local_state);
        //printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

        //getNewState2<MemoryAccess>(state,wt,P,W,inds,offsets,steps[threadIdx.x + count_batch * blockDim.x]);

        if( state == -1 )
            break;

        // Tally
        tallyContribution(entry,coeffs[stage]*wt*rhs[state],x);

        //count_batch++;

        // Check weight cutoff
        if( std::abs(wt/init_wt) < wt_cutoff )
            break;
   
    }

    // Store rng state back to global
    rng_state[tid] = local_state;
}


template<class MemoryAccess>
__global__ void run_forward_monte_carlo2(int N, int history_length, double wt_cutoff,
        int entry_histories, 
        const double * const H,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
        const double * const coeffs,
              double * const x,
        const double * const rhs, 
              hiprandState   *rng_state)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

//    extern __shared__ double sol[];

//    sol[threadIdx.x]=0.0;

    if( tid < N )
    {
        int entry=tid;
        
        for(int i=0; i<entry_histories; ++i)
        {
	    int state = entry;
	    double wt = 1.0;

	    hiprandState local_state = rng_state[tid];
	 
	    //printf("Starting history in state %i with weight %7.3f\n",state,wt);
	    if( state >= N )
	    {
		rng_state[tid] = local_state;
		return;
	    }
	    double init_wt = wt;

	    int stage = 0;

	    // Perform initial tally
	    tallyContribution(state,coeffs[stage]*wt*rhs[state],x);

            //tallyContribution2(coeffs[stage]*wt*rhs[state],&sol[threadIdx.x]);

	    for(; stage<=history_length; ++stage )
	    {
		// Move to new state
		getNewState<MemoryAccess>(state,wt,P,W,inds,offsets,&local_state);
		//printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

		if( state == -1 )
		    break;

		// Tally
		tallyContribution(entry,coeffs[stage]*wt*rhs[state],x);
                //tallyContribution2(coeffs[stage]*wt*rhs[state],&sol[threadIdx.x]);

		// Check weight cutoff
		if( std::abs(wt/init_wt) < wt_cutoff )
		    break;
	   
	    }

	    // Store rng state back to global
	    rng_state[tid] = local_state;
        }
    
        //x[entry]=sol[threadIdx.x];
        //v[entry]=sol[threaddx.x]*sol[threadIdx.x];

    }
}


template<class MemoryAccess>
__global__ void run_forward_monte_carlo3(int N, 
        int history_length, 
        double wt_cutoff,
        int entry_histories, 
        device_row_data* data, 
        const double * const coeffs,
        const int    * const offsets,
              double * const x,
        const double * const rhs, 
              hiprandState   *rng_state)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if( tid < N )
    {
        int entry=tid;
        
        for(int i=0; i<entry_histories; ++i)
        {
	    int state = entry;
	    double wt = 1.0;

	    hiprandState local_state = rng_state[tid];
	 
	    //printf("Starting history in state %i with weight %7.3f\n",state,wt);
	    if( state >= N )
	    {
		rng_state[tid] = local_state;
		return;
	    }
	    double init_wt = wt;

	    int stage = 0;

	    // Perform initial tally
	    tallyContribution(state,coeffs[stage]*wt*rhs[state],x);

            //tallyContribution2(coeffs[stage]*wt*rhs[state],&sol[threadIdx.x],&var[threadIdx.x], variance);

	    for(; stage<=history_length; ++stage )
	    {
		// Move to new state
		getNewState<MemoryAccess>(state,wt,data,offsets,&local_state);
		//printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

		if( state == -1 )
		    break;

		// Tally
		tallyContribution(entry,coeffs[stage]*wt*rhs[state],x);
                //tallyContribution2(coeffs[stage]*wt*rhs[state],&sol[threadIdx.x], &var[threadIdx.x], variance);

		// Check weight cutoff
		if( std::abs(wt/init_wt) < wt_cutoff )
		    break;
	   
	    }

	    // Store rng state back to global
	    rng_state[tid] = local_state;
        }
    
        //x[entry]=sol[threadIdx.x];

    }
}

        
//---------------------------------------------------------------------------//
/*!
 * \brief Constructor
 *
 * \param P Views into entries of probability matrix
 * \param W Views into entries of weight matrix
 * \param inds Views into nonzeros indices
 * \param offsets Starting indices for each matrix row
 * \param coeffs Polynomial coefficients
 * \param pl Problem parameters
 */
//---------------------------------------------------------------------------//
ForwardMcCuda::ForwardMcCuda(
        Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view     coeffs,
        Teuchos::RCP<Teuchos::ParameterList> pl)

  : d_N(mc_data->getIterationMatrix()->getGlobalNumRows())
{
    // Get parameters
    d_num_histories        = pl->get("num_histories",1000);
    d_max_history_length   = coeffs.size()-1;
    d_weight_cutoff        = pl->get("weight_cutoff",0.0);
    d_compute_variance     = pl->get("compute_variance", 0);
    d_struct               = pl->get("struct_matrix", 0);
    d_use_ldg              = pl->get("use_ldg", 0);
    d_device_number        = pl->get("device_number", 0);
    d_use_thread_per_entry = pl->get("thread_per_entry",0);
    std::string seed_type  = pl->get("seed_type", std::string("same"));

    VALIDATE( d_struct == 0 || d_struct == 1, 
            "Value for the flag to manage matrix data not valid" );
           
    VALIDATE( d_use_ldg==0 || d_use_ldg==1, 
            "Value for the texture memory handling not valid" );                    
    VALIDATE( d_use_thread_per_entry==0 || d_use_thread_per_entry==1,
            "Value for the task distribution between threads not valid" );

    VALIDATE( seed_type == std::string("same") 
              || seed_type == std::string("different")
              || seed_type == std::string("random"), 
              "Type of seed selected is not valid" );	

    int device_count = -1; 
    hipError_t e = hipGetDeviceCount( &device_count );
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(d_device_number <= device_count, 
            "The number of the device inserted exceeds the set of devices available");

    if( seed_type.c_str()==std::string("same") )   
    	d_seed_type = SEED_TYPE::SAME;
    else if( seed_type.c_str()==std::string("different") )
        d_seed_type = SEED_TYPE::DIFF;
    else if( seed_type.c_str()==std::string("random") ) 
        d_seed_type = SEED_TYPE::RAND;

    // Determine type of tally
    std::string estimator = pl->get<std::string>("estimator",
                                                 "collision");

    VALIDATE(estimator == "collision",
             "Only collision estimator is available.");

    // Should we print anything to screen
    std::string verb = profugus::to_lower(pl->get("verbosity","low"));
    if( verb == "none" )
        d_verbosity = NONE;
    else if( verb == "low" )
        d_verbosity = LOW;
    else if( verb == "high" )
        d_verbosity = HIGH;

    e = hipSetDevice( d_device_number );
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    prepareDeviceData(mc_data,coeffs);

    d_num_curand_calls = 0;
    d_rng_seed = pl->get<int>("rng_seed",1234);
 
}

//---------------------------------------------------------------------------//
// Solve problem using Monte Carlo
//---------------------------------------------------------------------------//
void ForwardMcCuda::solve(const MV &b, MV &x)
{
    Teuchos::ArrayRCP<const double> b_data = b.getData(0);
    thrust::device_vector<double> rhs(d_N);

    for(int i=0; i<d_N; ++i)
       rhs[i]=b_data[i];

    const double * const rhs_ptr = thrust::raw_pointer_cast(rhs.data());

    const double * H;
    const double * P;
    const double * W;
    const int    * inds;
    device_row_data * data_ptr;

    if( d_struct==0 )
    {
    	H       = thrust::raw_pointer_cast(d_H.data());
    	P       = thrust::raw_pointer_cast(d_P.data());
    	W       = thrust::raw_pointer_cast(d_W.data());
    	inds    = thrust::raw_pointer_cast(d_inds.data());
    }
    else
    	 data_ptr = thrust::raw_pointer_cast(mat_data.data());   

    const int    * const offsets = thrust::raw_pointer_cast(d_offsets.data());
    const double * const coeffs  = thrust::raw_pointer_cast(d_coeffs.data());

    // Create vector for state
    thrust::device_vector<double> x_vec(d_N);
    double * const x_ptr = thrust::raw_pointer_cast(x_vec.data());

    int block_size;
    int num_blocks;
    int tot_histories;
    
    if(d_use_thread_per_entry)
    {
	//instantiation of as many threads as the number of entries in the solution
    	block_size = std::min(256, d_N);
    	num_blocks = d_N / block_size + 1;
    }
    else 
    {
	//instiantiation of as many threads as the total number of histories
        tot_histories = d_num_histories * d_N;
        block_size = std::min(256,tot_histories);
        num_blocks = tot_histories / block_size + 1;    
    }

    VALIDATE( num_blocks > 0, "The size of the problem is too small" );

    hiprandState *rng_states;
    hipError_t e = hipMalloc((void **)&rng_states,
        block_size*num_blocks*sizeof(hiprandState));

    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to allocate memory");

    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    if( d_seed_type==SEED_TYPE::SAME )
    {
        std::cout<<"Same seed instantiated for all the threads"<<std::endl;

    	// Initialize RNG
    	SameSeed seed(d_rng_seed);
	initialize_rng<SameSeed><<<num_blocks,block_size>>>(rng_states,
    	    d_num_curand_calls, seed);
    }
    else if ( d_seed_type==SEED_TYPE::DIFF )
    {
        std::cout<<"Different adjacent seeds instantiated"<<std::endl;

    	DifferentSeed seed( block_size*num_blocks, d_rng_seed);
    	
    	initialize_rng<DifferentSeed><<<num_blocks, block_size>>>(rng_states,  
            d_num_curand_calls, seed);
    }
    else if ( d_seed_type==SEED_TYPE::RAND )
    {
        std::cout<<"Different random seeds instantiated from 0 to "<<
         RAND_MAX<<std::endl;

    	RandomSeed seed( block_size*num_blocks);
 
    	initialize_rng<RandomSeed><<<num_blocks, block_size>>>(rng_states,
            d_num_curand_calls, seed);
    }

    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to initialize RNG");
    d_num_curand_calls++;

    if( d_struct==0 )
    {
	if( d_use_thread_per_entry )
        {
            if( d_use_ldg==0 )	
            {   
		    run_forward_monte_carlo2<StandardAccess><<< num_blocks,block_size,sizeof(double)*block_size >>>(d_N,
		    	d_max_history_length, d_weight_cutoff, d_num_histories,
			H,P,W,inds,offsets,coeffs,x_ptr,rhs_ptr, rng_states);  
            }
            else
            {
		    run_forward_monte_carlo2<LDGAccess><<< num_blocks,block_size,sizeof(double)*block_size >>>(d_N,
		    	d_max_history_length, d_weight_cutoff, d_num_histories,
			H,P,W,inds,offsets,coeffs,x_ptr,rhs_ptr, rng_states);  
            }		
        }
	else    
        {
	    int batch_size = 5;
		       
            if( d_use_ldg==0 )		      
            {
		    run_forward_monte_carlo<StandardAccess><<< num_blocks,block_size >>>(d_N,
			d_max_history_length, d_weight_cutoff, d_num_histories, batch_size,
			H,P,W,inds,offsets,coeffs,x_ptr,rhs_ptr, rng_states);            
            }
            else
            {
		    run_forward_monte_carlo<LDGAccess><<< num_blocks,block_size >>>(d_N,
			d_max_history_length, d_weight_cutoff, d_num_histories, batch_size,
			H,P,W,inds,offsets,coeffs,x_ptr,rhs_ptr, rng_states);                   
            }
	}
    }	
    else
    {	
        if( d_use_ldg==0 )
        {
		run_forward_monte_carlo3<StandardAccess><<< num_blocks,block_size >>>( d_N,d_max_history_length,d_weight_cutoff,d_num_histories,data_ptr,
		        coeffs,offsets,x_ptr,rhs_ptr,rng_states  );
        }
        else
        {
		run_forward_monte_carlo3<LDGAccess><<< num_blocks,block_size >>>( d_N,d_max_history_length,d_weight_cutoff,d_num_histories,data_ptr,
		        coeffs,offsets,x_ptr,rhs_ptr,rng_states  );
        }
    }           
    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to execute MC kernel"); 

    // Copy data back to host
    //{
        Teuchos::ArrayRCP<double> x_data = x.getDataNonConst(0);
        thrust::copy(x_vec.begin(),x_vec.end(),x_data.get());
    //}

    // Scale by history count
    for( auto itr= x_data.begin(); itr != x_data.end(); ++itr )
        *itr /= static_cast<double>(d_num_histories); 

    // Free RNG state
    e = hipFree(rng_states);
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to deallocate memory");

}

//---------------------------------------------------------------------------//
// PRIVATE FUNCTIONS
//---------------------------------------------------------------------------//

//---------------------------------------------------------------------------//
// Extract matrices into ArrayView objects for faster data access
//---------------------------------------------------------------------------//

void ForwardMcCuda::prepareDeviceData(Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view coeffs)
{
    	Teuchos::RCP<const MATRIX> H = mc_data->getIterationMatrix();
    	Teuchos::RCP<const MATRIX> P = mc_data->getProbabilityMatrix();
    	Teuchos::RCP<const MATRIX> W = mc_data->getWeightMatrix();

    	d_offsets.resize(d_N+1);

	if(d_struct == 0)
	{
   		d_nnz = H->getNodeNumEntries();
    		d_H.resize(d_nnz);
    		d_P.resize(d_nnz);
    		d_W.resize(d_nnz);
    		d_inds.resize(d_nnz);
	
    		Teuchos::ArrayView<const double> val_row;
    		Teuchos::ArrayView<const int>    ind_row;
    		auto h_iter   = d_H.begin();
    		auto p_iter   = d_P.begin();
    		auto w_iter   = d_W.begin();
    		auto ind_iter = d_inds.begin();
    // This loop should perhaps be rewritten, right now a separate call
    // to hipMemcpy is performed for each row of each matrix
    // It might be more efficient to create a single vector on the CPU
    // and do a single copy to device?
    		d_offsets[0] = 0;
    		for( int i=0; i<d_N; ++i )
    		{
        		// Extract row i of matrix
        		H->getLocalRowView(i,ind_row,val_row);
        		thrust::copy(val_row.begin(),val_row.end(),h_iter);
        		h_iter += val_row.size();
        		P->getLocalRowView(i,ind_row,val_row);
        		thrust::copy(val_row.begin(),val_row.end(),p_iter);
        		p_iter += val_row.size();
       			W->getLocalRowView(i,ind_row,val_row);
        		thrust::copy(val_row.begin(),val_row.end(),w_iter);
        		w_iter += val_row.size();
        		thrust::copy(ind_row.begin(),ind_row.end(),ind_iter);
        		ind_iter += ind_row.size();
        		d_offsets[i+1] = d_offsets[i] + ind_row.size();
    		}
    		CHECK( h_iter   == d_H.end() );
    		CHECK( p_iter   == d_P.end() );
    		CHECK( w_iter   == d_W.end() );
    		CHECK( ind_iter == d_inds.end() );

    		// Copy coefficients into device vector
    		const_scalar_view::HostMirror coeffs_host = Kokkos::create_mirror_view(coeffs);
    		Kokkos::deep_copy(coeffs_host,coeffs);
    		d_coeffs.resize(coeffs.size());
    		thrust::copy(coeffs_host.ptr_on_device(),
                	 coeffs_host.ptr_on_device()+coeffs_host.size(),
                 	d_coeffs.begin());
	}

	else
	{
   		d_nnz = H->getNodeNumEntries();
    		Teuchos::RCP<const MATRIX> H = mc_data->getIterationMatrix();
    		Teuchos::RCP<const MATRIX> P = mc_data->getProbabilityMatrix();
    		Teuchos::RCP<const MATRIX> W = mc_data->getWeightMatrix();

    		Teuchos::ArrayView<const double> pval_row;
    		Teuchos::ArrayView<const double> hval_row;
    		Teuchos::ArrayView<const double> wval_row;
    		Teuchos::ArrayView<const int>    ind_row;
    // This loop should perhaps be rewritten, right now a separate call
    // to hipMemcpy is performed for each row of each matrix
    // It might be more efficient to create a single vector on the CPU
    // and do a single copy to device?
    		d_offsets[0] = 0;

    		thrust::host_vector< device_row_data > data_host( d_nnz );
    		mat_data.resize( d_nnz );

    		int count = 0;
    		for( int i=0; i<d_N; ++i )
    		{
        		// Extract row i of matrix
        		H->getLocalRowView(i,ind_row,hval_row);
        		P->getLocalRowView(i,ind_row,pval_row);
        		W->getLocalRowView(i,ind_row,wval_row); 
     
        		for( int j = 0; j < ind_row.size(); ++j )
        		{
            			data_host[count].H = hval_row[j];            
            			data_host[count].P = pval_row[j];
            			data_host[count].W = wval_row[j];
            			data_host[count].inds = ind_row[j];
            			count++;
        		}
        
        		d_offsets[i+1] = d_offsets[i] + ind_row.size();
    		}

    		CHECK( count == d_nnz );

    		thrust::copy( data_host.begin(), data_host.end(), mat_data.begin() );

    		// Copy coefficients into device vector
    		const_scalar_view::HostMirror coeffs_host = Kokkos::create_mirror_view(coeffs);
    		Kokkos::deep_copy(coeffs_host,coeffs);
    		d_coeffs.resize(coeffs.size());
    		thrust::copy(coeffs_host.ptr_on_device(),
                 	coeffs_host.ptr_on_device()+coeffs_host.size(),
                 	d_coeffs.begin());

        }
}


} // namespace alea
