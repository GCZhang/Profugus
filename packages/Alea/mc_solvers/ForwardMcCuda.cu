#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   ForwardMcCuda.cu
 * \author Steven Hamilton
 * \brief  Perform single history of adjoint MC
 */
//---------------------------------------------------------------------------//

#include <iterator>
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/generate.h>
#include <thrust/random.h>

#include "ForwardMcCuda.hh"
#include "utils/String_Functions.hh"
#include "harness/Warnings.hh"

namespace alea
{


//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//
__device__ void tallyContribution(int state, double wt, double * const x)
{
        // Collision estimator just adds weight
        atomicAdd(x+state,wt);
}

__device__ void tallyContribution2(double wt, double * const x)
{
        atomic_Add(x,wt);
}


//---------------------------------------------------------------------------//
/*!
 * \brief Tally contribution into vector
 */
//---------------------------------------------------------------------------//
__global__ void run_forward_monte_carlo(int N, int history_length, double wt_cutoff,
        int entry_histories, 
        int batch_size,
        const double * const H,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
        const double * const coeffs,
              double * const x,
        const double * const rhs, 
              hiprandState   *rng_state)
{
    int state = -1;
    double wt = 1.0;

    // Store rng state locally
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int entry = tid / entry_histories;

    if(entry >= N)
      return;

    state = entry;
    hiprandState local_state = rng_state[tid];
 
/*    extern __shared__ double steps[];
 
    for (int i = 0; i<batch_size; ++i)
        steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);

*/


    //initializeHistory2(state,wt,N,start_cdf,start_wt,steps[threadIdx.x]);

    //printf("Starting history in state %i with weight %7.3f\n",state,wt);
    if( state >= N )
    {
        rng_state[tid] = local_state;
        return;
    }
    double init_wt = wt;

    int stage = 0;

    // Perform initial tally
    tallyContribution(state,coeffs[stage]*wt*rhs[state],x);

  //  int count_batch = 0;

    for(; stage<=history_length; ++stage )
    {
/*        if (count_batch == batch_size)
        {

          //__syncthreads();
         count_batch = 0;
         for (int i = 0; i<batch_size; ++i)
            steps[threadIdx.x + i*blockDim.x] = hiprand_uniform_double(&local_state);
        }

*/

        // Move to new state
        getNewState(state,wt,P,W,inds,offsets,&local_state);
        //printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

        //getNewState2(state,wt,P,W,inds,offsets,steps[threadIdx.x + count_batch * blockDim.x]);

        if( state == -1 )
            break;

        // Tally
        tallyContribution(entry,coeffs[stage]*wt*rhs[state],x);

        //count_batch++;

        // Check weight cutoff
        if( std::abs(wt/init_wt) < wt_cutoff )
            break;
   
    }

    // Store rng state back to global
    rng_state[tid] = local_state;
}


__global__ void run_forward_monte_carlo2(int N, int history_length, double wt_cutoff,
        int entry_histories, 
        const double * const H,
        const double * const P,
        const double * const W,
        const int    * const inds,
        const int    * const offsets,
        const double * const coeffs,
              double * const x,
        const double * const rhs, 
              hiprandState   *rng_state)
{
    int state = -1;
    double wt = 1.0;

    // Store rng state locally
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ double sol[];    

    for (int i = 0; i<entry_histories; ++i)
        sol[threadIdx.x + i] = 0.0;

    if( tid < N )
    {
    	int entry = tid;

    	state = entry;
    	hiprandState local_state = rng_state[tid];
  
        for (int i = 0; i<entry_histories; ++i)
        {
    		//initializeHistory2(state,wt,N,start_cdf,start_wt,steps[threadIdx.x]);

    		//printf("Starting history in state %i with weight %7.3f\n",state,wt);
    		if( state >= N )
    		{
        		rng_state[tid] = local_state;
        		return;
    		}
    		double init_wt = wt;

    		int stage = 0;

    		// Perform initial tally
    		tallyContribution2(coeffs[stage]*wt*rhs[state],&sol[threadIdx.x + i]);

  	//  	int count_batch = 0;

    		for(; stage<=history_length; ++stage )
    		{
        		// Move to new state
        		getNewState(state,wt,P,W,inds,offsets,&local_state);
        		//printf("Stage %i, moving to state %i with new weight of %7.3f\n",stage,state,wt);

        		//getNewState2(state,wt,P,W,inds,offsets,steps[threadIdx.x + count_batch * blockDim.x]);

        		if( state == -1 )
            			break;

        		// Tally
        		tallyContribution2(coeffs[stage]*wt*rhs[state],&sol[threadIdx.x + i]);

        		// Check weight cutoff
        		if( std::abs(wt/init_wt) < wt_cutoff )
            			break;
   
    		} 
        }
        double update = 0.0;

        for (int i = 0; i< entry_hi
        
        
        stories; ++i)
            update += sol[threadIdx.x + i];

        update /= entry_histories;
        x[tid]=update;

    	// Store rng state back to global
    	rng_state[tid] = local_state;
    }
}



        
//---------------------------------------------------------------------------//
/*!
 * \brief Constructor
 *
 * \param P Views into entries of probability matrix
 * \param W Views into entries of weight matrix
 * \param inds Views into nonzeros indices
 * \param offsets Starting indices for each matrix row
 * \param coeffs Polynomial coefficients
 * \param pl Problem parameters
 */
//---------------------------------------------------------------------------//
ForwardMcCuda::ForwardMcCuda(
        Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view     coeffs,
        Teuchos::RCP<Teuchos::ParameterList> pl)

  : d_N(mc_data->getIterationMatrix()->getGlobalNumRows())
{
    // Get parameters
    d_num_histories      = pl->get("num_histories",1000);
    d_max_history_length = coeffs.size()-1;
    d_weight_cutoff      = pl->get("weight_cutoff",0.0);

    // Determine type of tally
    std::string estimator = pl->get<std::string>("estimator",
                                                 "collision");

    VALIDATE(estimator == "collision",
             "Only collision estimator is available.");

    // Should we print anything to screen
    std::string verb = profugus::to_lower(pl->get("verbosity","low"));
    if( verb == "none" )
        d_verbosity = NONE;
    else if( verb == "low" )
        d_verbosity = LOW;
    else if( verb == "high" )
        d_verbosity = HIGH;

    prepareDeviceData(mc_data,coeffs);

    d_num_curand_calls = 0;
    d_rng_seed = pl->get<int>("rng_seed",1234);
}

//---------------------------------------------------------------------------//
// Solve problem using Monte Carlo
//---------------------------------------------------------------------------//
void ForwardMcCuda::solve(const MV &b, MV &x)
{
    Teuchos::ArrayRCP<const double> b_data = b.getData(0);
    thrust::device_vector<double> rhs(d_N);

    for(int i=0; i<d_N; ++i)
       rhs[i]=b_data[i];

    const double * const rhs_ptr = thrust::raw_pointer_cast(rhs.data());

    const double * const H       = thrust::raw_pointer_cast(d_H.data());
    const double * const P       = thrust::raw_pointer_cast(d_P.data());
    const double * const W       = thrust::raw_pointer_cast(d_W.data());
    const int    * const inds    = thrust::raw_pointer_cast(d_inds.data());
    const int    * const offsets = thrust::raw_pointer_cast(d_offsets.data());
    const double * const coeffs  = thrust::raw_pointer_cast(d_coeffs.data());

    // Create vector for state
    thrust::device_vector<double> x_vec(d_N);
    double * const x_ptr = thrust::raw_pointer_cast(x_vec.data());

    //instiantiation of as many threads as the total number of histories
/*    int tot_histories = d_num_histories * d_N;

    int block_size = std::min(256,tot_histories);
    int num_blocks = tot_histories / block_size + 1;
    
    int block_size = std::min(256, tot_histories);
*/  

    //instantiation of as many threads as the number of entries in the solution
    int block_size = std::min(256, d_N);
    int num_blocks = d_N / block_size + 1;

     VALIDATE( num_blocks > 0, "The size of the problem is too small" );

    hiprandState *rng_states;
    hipError_t e = hipMalloc((void **)&rng_states,
        block_size*num_blocks*sizeof(hiprandState));

    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to allocate memory");

    // Initialize RNG
    //initialize_rng<<<num_blocks,block_size>>>(rng_states,d_rng_seed,
    //    d_num_curand_calls);

    thrust::device_vector<int> seeds( block_size*num_blocks);
    thrust::sequence(seeds.begin(), seeds.end(), d_rng_seed);
    int* seed_ptr = thrust::raw_pointer_cast(seeds.data());

    initialize_rng2<<<num_blocks, block_size>>>(rng_states, seed_ptr, 
          d_num_curand_calls);

    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to initialize RNG");
    d_num_curand_calls++;
   
//    run_forward_monte_carlo<<< num_blocks,block_size>>>(d_N,d_max_history_length, d_weight_cutoff, d_num_histories, batch_size,
//        H,P,W,inds,offsets,coeffs,x_ptr, rhs_ptr, rng_states);

    run_forward_monte_carlo2<<< num_blocks,block_size, d_num_histories*block_size>>>(d_N,d_max_history_length, d_weight_cutoff, d_num_histories,
        H,P,W,inds,offsets,coeffs,x_ptr, rhs_ptr, rng_states);


    // Check for errors in kernel launch
    e = hipGetLastError();
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to execute MC kernel"); 

    // Scale by history count
    /*for( auto itr= x_vec.begin(); itr != x_vec.end(); ++itr )
        *itr /= static_cast<double>(d_num_histories);*/

    // Copy data back to host
    {
        Teuchos::ArrayRCP<double> x_data = x.getDataNonConst(0);
        thrust::copy(x_vec.begin(),x_vec.end(),x_data.get());
    }

    // Free RNG state
    e = hipFree(rng_states);
    if( hipSuccess != e )
        std::cout << "Cuda Error: " << hipGetErrorString(e) << std::endl;

    VALIDATE(hipSuccess==e,"Failed to deallocate memory");
}

//---------------------------------------------------------------------------//
// PRIVATE FUNCTIONS
//---------------------------------------------------------------------------//

//---------------------------------------------------------------------------//
// Extract matrices into ArrayView objects for faster data access
//---------------------------------------------------------------------------//
void ForwardMcCuda::prepareDeviceData(Teuchos::RCP<const MC_Data> mc_data,
        const const_scalar_view coeffs)
{
    Teuchos::RCP<const MATRIX> H = mc_data->getIterationMatrix();
    Teuchos::RCP<const MATRIX> P = mc_data->getProbabilityMatrix();
    Teuchos::RCP<const MATRIX> W = mc_data->getWeightMatrix();

    d_nnz = H->getNodeNumEntries();
    d_H.resize(d_nnz);
    d_P.resize(d_nnz);
    d_W.resize(d_nnz);
    d_inds.resize(d_nnz);
    d_offsets.resize(d_N+1);

    Teuchos::ArrayView<const double> val_row;
    Teuchos::ArrayView<const int>    ind_row;
    auto h_iter   = d_H.begin();
    auto p_iter   = d_P.begin();
    auto w_iter   = d_W.begin();
    auto ind_iter = d_inds.begin();
    // This loop should perhaps be rewritten, right now a separate call
    // to hipMemcpy is performed for each row of each matrix
    // It might be more efficient to create a single vector on the CPU
    // and do a single copy to device?
    d_offsets[0] = 0;
    for( int i=0; i<d_N; ++i )
    {
        // Extract row i of matrix
        H->getLocalRowView(i,ind_row,val_row);
        thrust::copy(val_row.begin(),val_row.end(),h_iter);
        h_iter += val_row.size();
        P->getLocalRowView(i,ind_row,val_row);
        thrust::copy(val_row.begin(),val_row.end(),p_iter);
        p_iter += val_row.size();
        W->getLocalRowView(i,ind_row,val_row);
        thrust::copy(val_row.begin(),val_row.end(),w_iter);
        w_iter += val_row.size();
        thrust::copy(ind_row.begin(),ind_row.end(),ind_iter);
        ind_iter += ind_row.size();
        d_offsets[i+1] = d_offsets[i] + ind_row.size();
    }
    CHECK( h_iter   == d_H.end() );
    CHECK( p_iter   == d_P.end() );
    CHECK( w_iter   == d_W.end() );
    CHECK( ind_iter == d_inds.end() );

    // Copy coefficients into device vector
    const_scalar_view::HostMirror coeffs_host = Kokkos::create_mirror_view(coeffs);
    Kokkos::deep_copy(coeffs_host,coeffs);
    d_coeffs.resize(coeffs.size());
    thrust::copy(coeffs_host.ptr_on_device(),
                 coeffs_host.ptr_on_device()+coeffs_host.size(),
                 d_coeffs.begin());
}


} // namespace alea
