#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   cuda_utils/test/Texture_Vector_Test_Kernel.cu
 * \author Seth R Johnson
 * \date   Sat Sep 21 11:59:41 2013
 * \brief  Texture_Vector_Test_Kernel kernel definitions.
 * \note   Copyright (C) 2013 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//
#include "../cuda_utils/Kernel_Header.cuh"

#include "Texture_Vector_Test_Kernel.cuh"

#include "../cuda_utils/CudaDBC.hh"

namespace cuda_utils
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
template<typename Arch_Switch, typename T>
__global__ void texture_vector_test_kernel(
        const Texture_Vector_Kernel<Arch_Switch, T> input,
        size_t size,
        T*  output)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (; i < size; i += blockDim.x * gridDim.x)
    {
        output[i] = input[i];
    }
}

//---------------------------------------------------------------------------//
// HOST INTERFACES
//---------------------------------------------------------------------------//

template<typename Arch_Switch, typename T>
void texture_vector_test(
        const Texture_Vector<Arch_Switch, T>& input,
              Device_Vector<Arch_Switch, T>&  output)
{
    REQUIRE(input.size() == output.size());

    // CHECK for prior launch failure
    CudaCall(hipGetLastError());

    unsigned int num_threads = 64;
    unsigned int num_blocks  = output.size() / 64;

    texture_vector_test_kernel<Arch_Switch, T>
#ifdef __NVCC__
        <<<num_blocks, num_threads>>>
#endif
        (
            input.data(),
            input.size(),
            output.data());

    // Wait until kernel is finished
    CudaInsist(hipDeviceSynchronize(), "Kernel execution error");
}

//---------------------------------------------------------------------------//
// INSTANTIATIONS
//---------------------------------------------------------------------------//
#ifdef __NVCC__
typedef ::cuda_utils::arch::Device Arch_t;
#else
typedef ::cuda_utils::arch::Host Arch_t;
#endif

template void texture_vector_test(
        const Texture_Vector<Arch_t, int>&,
              Device_Vector<Arch_t, int>&);
template void texture_vector_test(
        const Texture_Vector<Arch_t, float>&,
              Device_Vector<Arch_t, float>&);
template void texture_vector_test(
        const Texture_Vector<Arch_t, double>&,
              Device_Vector<Arch_t, double>&);


//---------------------------------------------------------------------------//
} // end namespace cuda_utils

//---------------------------------------------------------------------------//
//                 end of cuda_utils/test/Texture_Vector_Test_Kernel.cu
//---------------------------------------------------------------------------//
