//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   cuda_utils/SerialDenseDeviceMatrix.cu
 * \author Stuart Slattery
 * \date   Tue Jan 5 2016
 * \brief  SerialDenseDeviceMatrix class definition.
 * \note   Copyright (C) 2014 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "SerialDenseDeviceMatrix.hh"

#include <hip/hip_runtime.h>

namespace cuda
{
//---------------------------------------------------------------------------//
// Size constructor.
SerialDenseDeviceMatrix::SerialDenseDeviceMatrix( const int num_rows,
						  const int num_cols,
						  const double fill_value )
    : d_num_rows( num_rows )
    , d_num_cols( num_cols )
    , d_owns_mem( true )
{
    Teuchos::TwoDArray<double> host_data( num_rows, num_cols, fill_value );
    allocate_and_copy( host_data );
}

//---------------------------------------------------------------------------//
// Host-data constructor.
SerialDenseDeviceMatrix::SerialDenseDeviceMatrix( 
    const int num_rows,
    const int num_cols,
    const Teuchos::Array<double>& host_data )
    : d_num_rows( num_rows )
    , d_num_cols( num_cols )
    , d_owns_mem( true )
{
    allocate_and_copy( host_data );
}

//---------------------------------------------------------------------------//
// Device data constructor.
SerialDenseDeviceMatrix::SerialDenseDeviceMatrix( const int num_rows, 
						  const int num_cols,
						  const bool owns_mem,
						  double* device_data )
    : d_num_rows( num_rows )
    , d_num_cols( num_cols )
    , d_owns_mem( owns_mem )
    , d_data( device_data )
{ /* ... */ }

//---------------------------------------------------------------------------//
// Destructor. Prohibits copy construction and assignment.
SerialDenseDeviceMatrix::~SerialDenseDeviceMatrix()
{
    if ( d_owns_mem )
    {
	hipFree( d_data );
    }
}

//---------------------------------------------------------------------------//
// Allocate and copy host data to the device.
void SerialDenseDeviceMatrix::allocate_and_copy( 
    const Teuchos::TwoDArray<double>& host_data )
{
    int mem_size = d_num_rows*d_num_cols*sizeof(double);
    hipMalloc( (void**) &d_data, mem_size );
    hipMemcpy( d_data, host_data.getRawPtr(), mem_size,
		hipMemcpyHostToDevice );
}

//---------------------------------------------------------------------------//

} // end namespace cuda

//---------------------------------------------------------------------------//
//                 end of SerialDenseDeviceMatrix.cu
//---------------------------------------------------------------------------//
