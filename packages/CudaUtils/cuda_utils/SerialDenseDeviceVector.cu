//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   cuda_utils/SerialDenseDeviceVector.cu
 * \author Stuart Slattery
 * \date   Tue Jan 5 2016
 * \brief  SerialDenseDeviceVector class definition.
 * \note   Copyright (C) 2014 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "SerialDenseDeviceVector.hh"

#include <hip/hip_runtime.h>

namespace cuda
{
//---------------------------------------------------------------------------//
// Size constructor.
SerialDenseDeviceVector::SerialDenseDeviceVector( const int size,
						  const double fill_value )
    : d_size( size )
    , d_owns_mem( true )
{
    Teuchos::Array<double> host_data( size, fill_value );
    allocate_and_copy( host_data );
}

//---------------------------------------------------------------------------//
// Host-data constructor.
SerialDenseDeviceVector::SerialDenseDeviceVector( 
    const Teuchos::Array<double>& host_data )
    : d_size( host_data.size() )
    , d_owns_mem( true )
{
    allocate_and_copy( host_data );
}

//---------------------------------------------------------------------------//
// Device data constructor.
SerialDenseDeviceVector::SerialDenseDeviceVector( const int size,
						  const bool owns_mem,
						  double* device_data )
    : d_size( size )
    , d_owns_mem( owns_mem )
    , d_data( device_data )
{ /* ... */ }

//---------------------------------------------------------------------------//
// Destructor. Prohibits copy construction and assignment.
SerialDenseDeviceVector::~SerialDenseDeviceVector()
{
    if ( d_owns_mem )
    {
	hipFree( d_data );
    }
}

//---------------------------------------------------------------------------//
// Allocate and copy host data to device.
void SerialDenseDeviceVector::allocate_and_copy( 
    const Teuchos::Array<double>& host_data )
{
    int mem_size = d_size*sizeof(double);
    hipMalloc( (void**) &d_data, mem_size );
    hipMemcpy( d_data, host_data.getRawPtr(), mem_size, 
		hipMemcpyHostToDevice );
}

//---------------------------------------------------------------------------//

} // end namespace cuda

//---------------------------------------------------------------------------//
//                 end of SerialDenseDeviceVector.cu
//---------------------------------------------------------------------------//
