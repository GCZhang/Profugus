#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   cuda_mc/Particle_Tester.cc
 * \author Steven Hamilton
 * \date   Wed Jan 20 14:57:16 2016
 * \brief  Particle_Tester member definitions.
 * \note   Copyright (C) 2016 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Particle_Tester.hh"
#include "../Particle.cuh"
#include "../../cuda_geometry/Mesh_Geometry.hh"

namespace cuda_mc
{

 __global__ void compute_randoms_kernel( double *randoms, int num_vals )
 {
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if( tid < num_vals )
     {
         // Create and initialize RNG state
         hiprandState_t rng_state;
         hiprand_init(tid,0,0,&rng_state);

         // Create particle and assign RNG
         Particle<cuda_profugus::Mesh_Geometry> p;
         p.set_rng(rng_state);

         // Generate random
         randoms[tid] = p.ran();
     }
}

__global__ void compute_groups_kernel( int *groups_in,
                                       int *groups_out,
                                       int num_vals )
{
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if( tid < num_vals )
     {
         // Create particle and assign group
         Particle<cuda_profugus::Mesh_Geometry> p;
         p.set_group(groups_in[tid]);

         // Generate random
         groups_out[tid] = p.group();
     }
}

__global__ void compute_matids_kernel( int *matids_in,
        int *matids_out,
        int num_vals )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < num_vals )
    {
        // Create particle and assign matid
        Particle<cuda_profugus::Mesh_Geometry> p;
        p.set_matid(matids_in[tid]);

        // Generate random
        matids_out[tid] = p.matid();
    }
}

void Particle_Tester::test_randoms( Vec_Dbl &host_rands )
{
    int num_vals = host_rands.size();

    typedef cuda::arch::Device Arch;
    cuda::Device_Vector<Arch,double> device_rands(num_vals);

    compute_randoms_kernel<<<1,num_vals>>>( device_rands.data(), num_vals );

    REQUIRE( hipGetLastError() == hipSuccess );

    device_rands.to_host(profugus::make_view(host_rands));
}

void Particle_Tester::test_groups( const Vec_Int &host_groups_in,
                                         Vec_Int &host_groups_out)
{
    int num_vals = host_groups_in.size();
    REQUIRE( host_groups_out.size() == num_vals );

    typedef cuda::arch::Device Arch;
    cuda::Device_Vector<Arch,int> device_groups_in(num_vals);
    cuda::Device_Vector<Arch,int> device_groups_out(num_vals);

    device_groups_in.assign(profugus::make_view(host_groups_in));

    compute_groups_kernel<<<1,num_vals>>>( device_groups_in.data(), 
                                           device_groups_out.data(),
                                           num_vals );

    REQUIRE( hipGetLastError() == hipSuccess );

    device_groups_out.to_host(profugus::make_view(host_groups_out));
}

void Particle_Tester::test_matids( const Vec_Int &host_matids_in,
                                         Vec_Int &host_matids_out)
{
    int num_vals = host_matids_in.size();
    REQUIRE( host_matids_out.size() == num_vals );

    typedef cuda::arch::Device Arch;
    cuda::Device_Vector<Arch,int> device_matids_in(num_vals);
    cuda::Device_Vector<Arch,int> device_matids_out(num_vals);

    device_matids_in.assign(profugus::make_view(host_matids_in));

    compute_matids_kernel<<<1,num_vals>>>( device_matids_in.data(), 
                                           device_matids_out.data(),
                                           num_vals );

    REQUIRE( hipGetLastError() == hipSuccess );

    device_matids_out.to_host(profugus::make_view(host_matids_out));
}
} // end namespace cuda_mc

//---------------------------------------------------------------------------//
//                 end of Particle_Tester.cc
//---------------------------------------------------------------------------//
