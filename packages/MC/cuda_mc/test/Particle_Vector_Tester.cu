#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   cuda_mc/test/Particle_Vector_Tester.cu
 * \author Stuart Slattery
 * \note   Copyright (C) 2013 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Particle_Vector_Tester.hh"

#include "cuda_utils/Hardware.hh"
#include "cuda_utils/CudaDBC.hh"

#include <hip/hip_runtime.h>

//---------------------------------------------------------------------------//
// CUDA Kernels
//---------------------------------------------------------------------------//
__global__ void ran_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			    double* ran )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    ran[i] = vector->ran( i );
}

//---------------------------------------------------------------------------//
__global__ void set_wt_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			       double wt )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_wt( i, wt );
}

//---------------------------------------------------------------------------//
__global__ void multiply_wt_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
				    double* wt )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->multiply_wt( i, wt[i] );
}

//---------------------------------------------------------------------------//
__global__ void wt_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			   double* wt )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    wt[i] = vector->wt( i );
}

//---------------------------------------------------------------------------//
__global__ void group_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			      int* group )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    group[i] = vector->group( i );
}

//---------------------------------------------------------------------------//
__global__ void set_group_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
				  int group )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_group( i, group );
}

//---------------------------------------------------------------------------//
__global__ void matid_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			      int* matid )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    matid[i] = vector->matid( i );
}

//---------------------------------------------------------------------------//
__global__ void set_matid_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
				  int matid )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_matid( i, matid );
}

//---------------------------------------------------------------------------//
__global__ void alive_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			      int* alive )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    alive[i] = vector->alive( i );
}

//---------------------------------------------------------------------------//
__global__ void live_kernel( Particle_Vector_Tester::Particle_Vector* vector )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->live( i );
}

//---------------------------------------------------------------------------//
__global__ void kill_kernel( Particle_Vector_Tester::Particle_Vector* vector )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->kill( i );
}

//---------------------------------------------------------------------------//
__global__ void set_event_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
				  typename Particle_Vector_Tester::Event_t* event )
{
    typename std::size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_event( i, event[i] );
}

//---------------------------------------------------------------------------//
__global__ void event_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			      typename Particle_Vector_Tester::Event_t* event )
{
    typename std::size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    event[i] = vector->event( i );
}

//---------------------------------------------------------------------------//
__global__ void set_geo_state_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
				      typename Particle_Vector_Tester::Geo_State_t geo_state )
{
    typename std::size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->geo_state( i ) = geo_state;
}

//---------------------------------------------------------------------------//
__global__ void geo_state_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
				  typename Particle_Vector_Tester::Geo_State_t* geo_state )
{
    typename std::size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    geo_state[i] = vector->geo_state( i );
}

//---------------------------------------------------------------------------//
__global__ void batch_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
			      int* batch )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    batch[i] = vector->batch( i );
}

//---------------------------------------------------------------------------//
__global__ void set_batch_kernel( Particle_Vector_Tester::Particle_Vector* vector, 
				  int batch )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_batch( i, batch );
}

//---------------------------------------------------------------------------//
// Particle_Vector_Tester
//---------------------------------------------------------------------------//
Particle_Vector_Tester::Particle_Vector_Tester( const int num_particle, 
						const profugus::RNG& rng )
    : d_size( num_particle )
{
    // Acquire hardware for the test.
    cuda::Hardware<cuda::arch::Device>::acquire();

    // Create the vector after hardware acquisition.
    d_vector = cuda::Shared_Device_Ptr<Particle_Vector>( num_particle, rng );
}

//---------------------------------------------------------------------------//
// get a vector of random numbers for the vector.
Teuchos::Array<double> Particle_Vector_Tester::ran()
{
    double* device_ran;
    hipMalloc( (void**) &device_ran, d_size * sizeof(double) );

    int num_block = 4;
    ran_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_ran );

    Teuchos::Array<double> host_ran( d_size );
    hipMemcpy( host_ran.getRawPtr(), device_ran, d_size * sizeof(double),
		hipMemcpyDeviceToHost );

    hipFree( device_ran );
    return host_ran;
}

//---------------------------------------------------------------------------//
// set the entire vector to the same weight.
void Particle_Vector_Tester::set_wt( const double wt )
{
    int num_block = 4;
    set_wt_kernel<<<num_block,d_size/num_block>>>( d_vector.get_device_ptr(), wt );
}

//---------------------------------------------------------------------------//
// mulitply each particle weight by a different value.
void Particle_Vector_Tester::multiply_wt( const Teuchos::Array<double>& wt )
{
    double* device_wt;
    hipMalloc( (void**) &device_wt, d_size * sizeof(double) );
    hipMemcpy( device_wt, wt.getRawPtr(), d_size * sizeof(double),
		hipMemcpyHostToDevice );
    
    int num_block = 4;
    multiply_wt_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_wt );

    hipFree( device_wt );
}

//---------------------------------------------------------------------------//
// get a vector of weights.
Teuchos::Array<double> Particle_Vector_Tester::wt()
{
    double* device_wt;
    hipMalloc( (void**) &device_wt, d_size * sizeof(double) );
    int num_block = 4;
    wt_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_wt );

    Teuchos::Array<double> host_wt( d_size );
    hipMemcpy( host_wt.getRawPtr(), device_wt, d_size * sizeof(double),
		hipMemcpyDeviceToHost );

    hipFree( device_wt );
    return host_wt;
}

//---------------------------------------------------------------------------//
// get a vector of groups.
Teuchos::Array<int> Particle_Vector_Tester::group()
{
    int* device_group;
    hipMalloc( (void**) &device_group, d_size * sizeof(int) );
    int num_block = 4;
    group_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_group );

    Teuchos::Array<int> host_group( d_size );
    hipMemcpy( host_group.getRawPtr(), device_group, d_size * sizeof(int),
		hipMemcpyDeviceToHost );

    hipFree( device_group );
    return host_group;
}

//---------------------------------------------------------------------------//
// Set the entire vector to a group.
void Particle_Vector_Tester::set_group( const int group )
{
    int num_block = 4;
    set_group_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), group );
}

//---------------------------------------------------------------------------//
// get a vector of matids.
Teuchos::Array<int> Particle_Vector_Tester::matid()
{
    int* device_matid;
    hipMalloc( (void**) &device_matid, d_size * sizeof(int) );
    int num_block = 4;
    matid_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_matid );

    Teuchos::Array<int> host_matid( d_size );
    hipMemcpy( host_matid.getRawPtr(), device_matid, d_size * sizeof(int),
		hipMemcpyDeviceToHost );

    hipFree( device_matid );
    return host_matid;
}

//---------------------------------------------------------------------------//
// Set the entire vector to a matid.
void Particle_Vector_Tester::set_matid( const int matid )
{
    int num_block = 4;
    set_matid_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), matid );
}

//---------------------------------------------------------------------------//
// get a vector of alive status.
Teuchos::Array<int> Particle_Vector_Tester::alive()
{
    int* device_alive;
    hipMalloc( (void**) &device_alive, d_size * sizeof(int) );
    int num_block = 4;
    alive_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_alive );

    Teuchos::Array<int> host_alive( d_size );
    hipMemcpy( host_alive.getRawPtr(), device_alive, d_size * sizeof(int),
		hipMemcpyDeviceToHost );

    hipFree( device_alive );
    return host_alive;
}

//---------------------------------------------------------------------------//
// set the whole vector to live.
void Particle_Vector_Tester::live()
{
    int num_block = 4;
    live_kernel<<<num_block,d_size/num_block>>>( d_vector.get_device_ptr() );
}

//---------------------------------------------------------------------------//
// kill the whole vector.
void Particle_Vector_Tester::kill()
{
    int num_block = 4;
    kill_kernel<<<num_block,d_size/num_block>>>( d_vector.get_device_ptr() );
}

//---------------------------------------------------------------------------//
// get a vector of events
Teuchos::Array<typename Particle_Vector_Tester::Event_t> 
Particle_Vector_Tester::event()
{
    Event_t* device_event;
    hipMalloc( (void**) &device_event, d_size * sizeof(Event_t) );
    int num_block = 4;
    event_kernel<<<num_block,d_size/num_block>>>(
	d_vector.get_device_ptr(), device_event );

    Teuchos::Array<Event_t> host_event( d_size );
    hipMemcpy( host_event.getRawPtr(), device_event, d_size * sizeof(Event_t),
		hipMemcpyDeviceToHost );

    hipFree( device_event );
    return host_event;
}

//---------------------------------------------------------------------------//
// set a vector of events
void Particle_Vector_Tester::set_event( const Teuchos::Array<Event_t>& events )
{
    Event_t* device_event;
    hipMalloc( (void**) &device_event, d_size * sizeof(Event_t) );
    hipMemcpy( device_event, events.getRawPtr(), d_size * sizeof(Event_t),
		hipMemcpyHostToDevice );
    
    int num_block = 4;
    set_event_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_event );

    hipFree( device_event );
}

//---------------------------------------------------------------------------//
// set the geometry state for the whole vector
void Particle_Vector_Tester::set_geo_state( const Geo_State_t& geo_state )
{
    int num_block = 4;
    set_geo_state_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), geo_state );
}

//---------------------------------------------------------------------------//
// get the geometry state for the whole vector
Teuchos::Array<typename Particle_Vector_Tester::Geo_State_t> 
Particle_Vector_Tester::geo_state()
{
    Geo_State_t* device_geo_state;
    hipMalloc( (void**) &device_geo_state, d_size * sizeof(Geo_State_t) );

    int num_block = 4;
    geo_state_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_geo_state );

    Teuchos::Array<Geo_State_t> host_geo_state( d_size );
    hipMemcpy( host_geo_state.getRawPtr(), device_geo_state, d_size * sizeof(Geo_State_t),
		hipMemcpyDeviceToHost );

    hipFree( device_geo_state );
    return host_geo_state;
}

//---------------------------------------------------------------------------//
// get a vector of batches.
Teuchos::Array<int> Particle_Vector_Tester::batch()
{
    int* device_batch;
    hipMalloc( (void**) &device_batch, d_size * sizeof(int) );

    int num_block = 4;
    batch_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), device_batch );

    Teuchos::Array<int> host_batch( d_size );
    hipMemcpy( host_batch.getRawPtr(), device_batch, d_size * sizeof(int),
		hipMemcpyDeviceToHost );

    hipFree( device_batch );
    return host_batch;
}

//---------------------------------------------------------------------------//
// Set the entire vector to a batch.
void Particle_Vector_Tester::set_batch( const int batch )
{
    int num_block = 4;
    set_batch_kernel<<<num_block,d_size/num_block>>>( 
	d_vector.get_device_ptr(), batch );
}

//---------------------------------------------------------------------------//
//                 end of cuda_mc/Particle_Vector_Tester.cu
//---------------------------------------------------------------------------//
