#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   cuda_mc/test/Collision_Tally_Tester.cu
 * \author Stuart Slattery
 * \note   Copyright (C) 2013 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Collision_Tally_Tester.hh"

#include "cuda_utils/Hardware.hh"
#include "cuda_utils/CudaDBC.hh"

#include <hip/hip_runtime.h>

//---------------------------------------------------------------------------//
// CUDA Kernels
//---------------------------------------------------------------------------//
__global__ 
void set_wt_kernel( Collision_Tally_Tester::Collision_Tally* vector, 
		    double* wt )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_wt( i, wt[i] );
}

//---------------------------------------------------------------------------//
__global__ 
void set_event_kernel( Collision_Tally_Tester::Collision_Tally* vector, 
		       typename Collision_Tally_Tester::Event_t* event )
{
    typename std::size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_event( i, event[i] );
}

//---------------------------------------------------------------------------//
__global__ 
void set_geo_state_kernel( 
    Collision_Tally_Tester::Collision_Tally* vector, 
    typename Collision_Tally_Tester::Geo_State_t* geo_state )
{
    typename std::size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->geo_state( i ) = geo_state[i];
}

//---------------------------------------------------------------------------//
__global__ 
void set_batch_kernel( Collision_Tally_Tester::Collision_Tally* vector, 
		       int* batch )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_batch( i, batch[i] );
}

//---------------------------------------------------------------------------//
__global__ 
void live_kernel( Particle_Vector_Tester::Particle_Vector* vector )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->live( i );
}

//---------------------------------------------------------------------------//
// Collision_Tally_Tester
//---------------------------------------------------------------------------//
Collision_Tally_Tester::Collision_Tally_Tester(  const std::vector<double>& x_edges,
						 const std::vector<double>& y_edges,
						 const std::vector<double>& z_edges,
						 const int num_particle, 
						 const profugus::RNG& rng,
						 const int num_batch )
{
    // Acquire hardware for the test.
    cuda::Hardware<cuda::arch::Device>::acquire();

    // Create the geometry.
    d_geometry = cuda::shared_device_ptr<Geometry>( x_edges, y_edges, z_edges );
    int num_cells = d_geometry.get_host_ptr()->num_cells();

    // Create the particle vector.
    d_vector = cuda::shared_device_ptr<Particle_Vector>( num_particle, rng );

    // Initialize the state of the particle vector. The first half of the
    // cells will have a particle with a collision event, the rest will
    // not. There will be one particle per cell per batch.

    // Cuda launch params.
    int num_threads = 256;
    int num_block = d_size / num_threads;

    // Set the weight. Weight assigned is the cell id.
    Teuchos::Array<double> wt( num_particle );
    for ( int b = 0; b < num_batch; ++b )
    {
	for ( int c = 0; c < num_cells; ++c )
	{
	    wt[ b*num_cells + c ] = c;
	}
    }
    double* device_wt;
    hipMalloc( (void**) &device_wt, d_size * sizeof(double) );
    hipMemcpy( device_wt, wt.getRawPtr(), d_size * sizeof(double),
		hipMemcpyHostToDevice );
    set_wt_kernel<<<num_block,num_threads>>>( 
	d_vector.get_device_ptr(), device_wt );
    hipFree( device_wt );

    // Set the event. If the particle is not in the first cells it does not
    // collide.
    Teuchos::Array<Event_t>& events( num_particle );
    for ( int b = 0; b < num_batch; ++b )
    {
	for ( int c = 0; c < num_cells; ++c )
	{
	    if ( c < num_cells / 2 )
	    {
		events[ b*num_cells + c ] = profugus::events::COLLISION;
	    }
	    else
	    {
		events[ b*num_cells + c ] = profugus::events::ABSORPTION;
	    }
	}
    }
    Event_t* device_event;
    hipMalloc( (void**) &device_event, d_size * sizeof(Event_t) );
    hipMemcpy( device_event, events.getRawPtr(), d_size * sizeof(Event_t),
		hipMemcpyHostToDevice );
    set_event_kernel<<<num_block,num_threads>>>( 
	d_vector.get_device_ptr(), device_event );
    hipFree( device_event );

    // Set the batch
    Teuchos::Array<int> batch( num_particle );
    for ( int b = 0; b < num_batch; ++b )
    {
	for ( int c = 0; c < num_cells; ++c )
	{
	    batch[ b*num_cells + c ] = b;
	}
    }
    int* device_batch;
    hipMalloc( (void**) &device_batch, d_size * sizeof(int) );
    hipMemcpy( device_batch, batch.getRawPtr(), d_size * sizeof(int),
		hipMemcpyHostToDevice );
    set_batch_kernel<<<num_block,num_threads>>>( 
	d_vector.get_device_ptr(), device_batch );
    hipFree( device_batch );

    // Set the geometry state. One particle per cell per batch.
    Teuchos::Array<Geo_State_t> geo_state;
    for ( int b = 0; b < num_geo_state; ++b )
    {
	for ( int c = 0; c < num_cells; ++c )
	{
	    int i, j, k;
	    d_geometry.get_host_ptr()->mesh().cardinal( c, i, j, k );
	    geo_state[ b*num_cells + c ].ijk.i = i;
	    geo_state[ b*num_cells + c ].ijk.j = j;
	    geo_state[ b*num_cells + c ].ijk.k = k;
	}
    }
    int* device_geo_state;
    hipMalloc( (void**) &device_geo_state, d_size * sizeof(int) );
    hipMemcpy( device_geo_state, geo_state.getRawPtr(), d_size * sizeof(int),
		hipMemcpyHostToDevice );
    set_geo_state_kernel<<<num_block,num_threads>>>( 
	d_vector.get_device_ptr(), device_geo_state );
    hipFree( device_geo_state );

    // Set the particles to alive.
    live_kernel<<<num_block,num_threads>>>( d_vector.get_device_ptr() );
}

//---------------------------------------------------------------------------//
//                 end of cuda_mc/Collision_Tally_Tester.cu
//---------------------------------------------------------------------------//
