#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   cuda_mc/test/Physics_Tester.cu
 * \author Stuart Slattery
 * \note   Copyright (C) 2013 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Physics_Tester.hh"

#include "cuda_utils/Hardware.hh"
#include "cuda_utils/CudaDBC.hh"
#include "cuda_utils/Utility_Functions.hh"
#include "cuda_utils/Memory.cuh"

#include <Teuchos_Array.hpp>

#include <hip/hip_runtime.h>

//---------------------------------------------------------------------------//
// CUDA Kernels
//---------------------------------------------------------------------------//
__global__ void geometry_initialize_kernel( 
    Physics_Tester::Particle_Vector* vector, 
    const Physics_Tester::Geometry* geometry,
    const Physics_Tester::Space_Vector r,
    const Physics_Tester::Space_Vector d,
    const int matid,
    const int num_particle )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < num_particle ) 
    {
	geometry->initialize( r, d, vector->geo_state(i) );
	vector->set_matid( i, matid );
	vector->set_event( i, cuda_profugus::events::COLLISION );
	vector->set_wt( i, 0.9 );
    }
}

//---------------------------------------------------------------------------//
__global__ void sample_group_kernel( 
    Physics_Tester::Particle_Vector* vector,
    const double* cdf,
    const int cdf_size,
    const int num_particle )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < num_particle ) 
    {
	int group =
	    cuda::utility::sample_discrete_CDF( cdf_size, cdf, vector->ran(i) );
	vector->set_group( i, group );
    }
}

//---------------------------------------------------------------------------//
// Physics_Tester
//---------------------------------------------------------------------------//
Physics_Tester::Physics_Tester( 
    const std::vector<double>& x_edges,
    const std::vector<double>& y_edges,
    const std::vector<double>& z_edges,
    const int vector_size,
    const profugus::RNG& rng,
    Teuchos::ParameterList& db,
    const profugus::XS& xs )
    : d_size( vector_size )
    , d_particle_tester( vector_size, rng )
{
    // Acquire hardware for the test.
    if ( !cuda::Hardware<cuda::arch::Device>::have_acquired() )
	cuda::Hardware<cuda::arch::Device>::acquire();

    // Create the geometry on the host.
    std::shared_ptr<Geometry> host_geom = 
	std::make_shared<Geometry>( x_edges, y_edges, z_edges );
    int num_cells = host_geom->num_cells();

    // Set matids with the geometry on the host.
    std::vector<typename Geometry::matid_type> matids( num_cells, 0 );
    host_geom->set_matids( matids );

    // Create a device copy of the geometry.
    d_geometry = cuda::Shared_Device_Ptr<Geometry>( host_geom );

    // Create the physics.
    d_physics = cuda::shared_device_ptr<Physics>( db, xs );

    // Set the geometry with the physics.
    d_physics.get_host_ptr()->set_geometry( d_geometry );
}

//---------------------------------------------------------------------------//
// Initialize particles with the geometry and set to collide.
void Physics_Tester::geometry_initialize( 
    const Space_Vector r, const Space_Vector d, const int matid )
{
    unsigned int threads_per_block = 
	cuda::Hardware<cuda::arch::Device>::num_cores_per_mp();
    unsigned int num_blocks = d_size / threads_per_block;
    if ( d_size % threads_per_block > 0 ) ++num_blocks;

    geometry_initialize_kernel<<<num_blocks,threads_per_block>>>(
	particles().get_device_ptr(), d_geometry.get_device_ptr(),
	r, d, matid, d_size );
}

//---------------------------------------------------------------------------//
// Sample a cdf and set the particle group.
void Physics_Tester::sample_group( const std::vector<double>& cdf )
{
    // copy the cdf to the device.
    double* device_cdf;
    cuda::memory::Malloc( device_cdf, cdf.size() );
    cuda::memory::Copy_To_Device( device_cdf, cdf.data(), cdf.size() );

    // Sample the cdf and set the particle groups.
    unsigned int threads_per_block = 
	cuda::Hardware<cuda::arch::Device>::num_cores_per_mp();
    unsigned int num_blocks = d_size / threads_per_block;
    if ( d_size % threads_per_block > 0 ) ++num_blocks;

    sample_group_kernel<<<num_blocks,threads_per_block>>>(
	particles().get_device_ptr(), device_cdf, cdf.size(), d_size );

    // free allocated data.
    cuda::memory::Free( device_cdf );
}

//---------------------------------------------------------------------------//
//                 end of cuda_mc/Physics_Tester.cu
//---------------------------------------------------------------------------//
