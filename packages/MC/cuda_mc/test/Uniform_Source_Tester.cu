#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   cuda_mc/Uniform_Source_Tester.cc
 * \author Steven Hamilton
 * \date   Wed Jan 20 14:57:16 2016
 * \brief  Uniform_Source_Tester member definitions.
 * \note   Copyright (C) 2016 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Uniform_Source_Tester.hh"
#include "../Uniform_Source.cuh"
#include "../Definitions.hh"
#include "../Source_Provider.cuh"

#include "utils/View_Field.hh"
#include "cuda_geometry/Mesh_Geometry.hh"

namespace cuda_mc
{

typedef cuda_utils::Space_Vector            Space_Vector;
typedef cuda_profugus::Mesh_Geometry        Geometry;
typedef cuda_mc::Uniform_Source<Geometry>   Uniform_Src;
typedef cuda_mc::Particle<Geometry>         Particle_t;
typedef cuda_mc::RNG_State_t                RNG_State;

__global__ void compute_source_kernel( Uniform_Src   source,
                                       Space_Vector *pts,
                                       Space_Vector *dirs,
                                       int           num_vals )
{
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if( tid < num_vals )
     {
         hiprandState_t rng;
         hiprand_init( 345, tid, 0, &rng );

         auto p = source.get_particle(tid,&rng);

         auto geo_state = p.geo_state();
         auto r = geo_state.d_r;
         pts[tid].x = r.x;
         pts[tid].y = r.y;
         pts[tid].z = r.z;

         auto omega = geo_state.d_dir;
         dirs[tid].x = omega.x;
         dirs[tid].y = omega.y;
         dirs[tid].z = omega.z;
     }
}

// Extract data from particles
__global__ void extract_data( Particle_t         *parts,
                              double             *x,
                              double             *y,
                              double             *z,
                              double             *mu,
                              double             *eta,
                              double             *xi,
                              int                 N )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < N )
    {
        auto p = parts[tid];

        auto geo_state = p.geo_state();
        auto r = geo_state.d_r;
        x[tid] = r.x;
        y[tid] = r.y;
        z[tid] = r.z;

        auto omega = geo_state.d_dir;
        mu[tid]  = omega.x;
        eta[tid] = omega.y;
        xi[tid]  = omega.z;
    }
}

void Uniform_Source_Tester::test_source( const Vec_Dbl &geom_bounds,
                                         const Vec_Dbl &src_bounds,
                                               Vec_Dbl &x_locs,
                                               Vec_Dbl &y_locs,
                                               Vec_Dbl &z_locs,
                                               Vec_Dbl &mu,
                                               Vec_Dbl &eta,
                                               Vec_Dbl &xi)
{
    // Copy values to device
    int Np = x_locs.size();
    REQUIRE( Np == y_locs.size() );
    REQUIRE( Np == z_locs.size() );
    REQUIRE( Np == mu.size() );
    REQUIRE( Np == eta.size() );
    REQUIRE( Np == xi.size() );
    thrust::device_vector<Space_Vector> device_pts(Np);
    thrust::device_vector<Space_Vector> device_dirs(Np);

    REQUIRE( geom_bounds.size() == 6 );

    // Build geometry edges, mesh doesn't really matter
    int Nx = 10;
    double xlo = geom_bounds[0];
    double xhi = geom_bounds[1];
    double dx = (xhi - xlo) / static_cast<double>(Nx);
    std::vector<double> x_edges(Nx+1,0.0);
    for( int ix = 0; ix < Nx+1; ++ix )
        x_edges[ix] = xlo + static_cast<double>(ix) * dx;

    int Ny = 10;
    double ylo = geom_bounds[2];
    double yhi = geom_bounds[3];
    double dy = (yhi - ylo) / static_cast<double>(Ny);
    std::vector<double> y_edges(Ny+1,0.0);
    for( int iy = 0; iy < Ny+1; ++iy )
        y_edges[iy] = ylo + static_cast<double>(iy) * dy;

    int Nz = 10;
    double zlo = geom_bounds[4];
    double zhi = geom_bounds[5];
    double dz = (zhi - zlo) / static_cast<double>(Nz);
    std::vector<double> z_edges(Nz+1,0.0);
    for( int iz = 0; iz < Nz+1; ++iz )
        z_edges[iz] = zlo + static_cast<double>(iz) * dz;

    // Build Geometry
    auto geom = cuda::shared_device_ptr<Geometry>(x_edges,y_edges,z_edges);

    auto db = Teuchos::rcp( new Teuchos::ParameterList() );
    db->set("Np",Np);
    db->set("num_groups",2);

    // Build box shape for source
    REQUIRE( src_bounds.size() == 6 );
    auto src_shape = cuda::shared_device_ptr<cuda_mc::Box_Shape>(
            src_bounds[0], src_bounds[1],
            src_bounds[2], src_bounds[3],
            src_bounds[4], src_bounds[5]);

    // Build source
    Uniform_Src source(db,geom);
    source.build_source(src_shape);

    // Launch kernel
    std::cout << "Launching kernel with " << Np << " particles" << std::endl;
    compute_source_kernel<<<1,Np>>>(source, device_pts.data().get(),
                                    device_dirs.data().get(), Np);
    REQUIRE( hipGetLastError() == hipSuccess );

    // Copy back to host
    std::vector<Space_Vector> host_pts(Np);
    std::vector<Space_Vector> host_dirs(Np);
    thrust::copy(device_pts.begin(),device_pts.end(),host_pts.begin());
    thrust::copy(device_dirs.begin(),device_dirs.end(),host_dirs.begin());

    for( int ip = 0; ip < Np; ++ip )
    {
        x_locs[ip] = host_pts[ip].x;
        y_locs[ip] = host_pts[ip].y;
        z_locs[ip] = host_pts[ip].z;
        mu[ip]     = host_dirs[ip].x;
        eta[ip]    = host_dirs[ip].y;
        xi[ip]     = host_dirs[ip].z;
    }
}

void Uniform_Source_Tester::test_host_api( const Vec_Dbl &geom_bounds,
                                           const Vec_Dbl &src_bounds,
                                                 Vec_Dbl &x_locs,
                                                 Vec_Dbl &y_locs,
                                                 Vec_Dbl &z_locs,
                                                 Vec_Dbl &mu,
                                                 Vec_Dbl &eta,
                                                 Vec_Dbl &xi)
{
    // Copy values to device
    int Np = x_locs.size();
    REQUIRE( Np == y_locs.size() );
    REQUIRE( Np == z_locs.size() );
    REQUIRE( Np == mu.size() );
    REQUIRE( Np == eta.size() );
    REQUIRE( Np == xi.size() );

    REQUIRE( geom_bounds.size() == 6 );

    // Build geometry edges, mesh doesn't really matter
    int Nx = 10;
    double xlo = geom_bounds[0];
    double xhi = geom_bounds[1];
    double dx = (xhi - xlo) / static_cast<double>(Nx);
    std::vector<double> x_edges(Nx+1,0.0);
    for( int ix = 0; ix < Nx+1; ++ix )
        x_edges[ix] = xlo + static_cast<double>(ix) * dx;

    int Ny = 10;
    double ylo = geom_bounds[2];
    double yhi = geom_bounds[3];
    double dy = (yhi - ylo) / static_cast<double>(Ny);
    std::vector<double> y_edges(Ny+1,0.0);
    for( int iy = 0; iy < Ny+1; ++iy )
        y_edges[iy] = ylo + static_cast<double>(iy) * dy;

    int Nz = 10;
    double zlo = geom_bounds[4];
    double zhi = geom_bounds[5];
    double dz = (zhi - zlo) / static_cast<double>(Nz);
    std::vector<double> z_edges(Nz+1,0.0);
    for( int iz = 0; iz < Nz+1; ++iz )
        z_edges[iz] = zlo + static_cast<double>(iz) * dz;

    // Build Geometry
    auto geom_host = std::make_shared<Geometry>(x_edges,y_edges,z_edges);
    std::vector<int> matids(Nx*Ny*Nz,0);
    geom_host->set_matids(matids);
    cuda::Shared_Device_Ptr<Geometry> geom(geom_host);

    auto db = Teuchos::rcp( new Teuchos::ParameterList() );
    db->set("Np",Np);
    db->set("num_groups",2);

    // Build box shape for source
    REQUIRE( src_bounds.size() == 6 );
    auto src_shape = cuda::shared_device_ptr<cuda_mc::Box_Shape>(
            src_bounds[0], src_bounds[1],
            src_bounds[2], src_bounds[3],
            src_bounds[4], src_bounds[5]);

    // Build source
    auto source_host = std::make_shared<Uniform_Src>(db,geom);
    source_host->build_source(src_shape);

    // Initialize RNG
    auto rng_control = std::make_shared<cuda_mc::RNG_Control>(1234);

    // Get vector of particles from source
    thrust::device_vector<Particle<Geometry>> particles;

    cuda_mc::Source_Provider<Geometry> provider;
    provider.get_particles(source_host,rng_control,particles);
    REQUIRE( particles.size() == source_host->num_to_transport() );

    // Extract data from particles
    thrust::device_vector<double> x_dev(   Np );
    thrust::device_vector<double> y_dev(   Np );
    thrust::device_vector<double> z_dev(   Np );
    thrust::device_vector<double> mu_dev(  Np );
    thrust::device_vector<double> eta_dev( Np );
    thrust::device_vector<double> xi_dev(  Np );

    extract_data<<<1,Np>>>( particles.data().get(),
                            x_dev.data().get(),
                            y_dev.data().get(),
                            z_dev.data().get(),
                            mu_dev.data().get(),
                            eta_dev.data().get(),
                            xi_dev.data().get(),
                            Np );

    // Copy back to host
    thrust::host_vector<double> x_host(Np);
    thrust::host_vector<double> y_host(Np);
    thrust::host_vector<double> z_host(Np);
    thrust::host_vector<double> mu_host(Np);
    thrust::host_vector<double> eta_host(Np);
    thrust::host_vector<double> xi_host(Np);
    x_host   = x_dev;
    y_host   = y_dev;
    z_host   = z_dev;
    mu_host  = mu_dev;
    eta_host = eta_dev;
    xi_host  = xi_dev;

    for( int ip = 0; ip < Np; ++ip )
    {
        x_locs[ip] = x_host[ip];
        y_locs[ip] = y_host[ip];
        z_locs[ip] = z_host[ip];
        mu[ip]     = mu_host[ip];
        eta[ip]    = eta_host[ip];
        xi[ip]     = xi_host[ip];
    }
}

} // end namespace cuda_mc

//---------------------------------------------------------------------------//
//                 end of Uniform_Source_Tester.cc
//---------------------------------------------------------------------------//
