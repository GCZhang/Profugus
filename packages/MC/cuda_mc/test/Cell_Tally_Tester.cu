#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   cuda_mc/Cell_Tally_Tester.cc
 * \author Steven Hamilton
 * \date   Wed Jan 20 16:13:24 2016
 * \brief  Cell_Tally_Tester member definitions.
 * \note   Copyright (C) 2016 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Cell_Tally_Tester.hh"
#include "../Cell_Tally.cuh"
#include "cuda_geometry/Mesh_Geometry.hh"
#include "CudaUtils/cuda_utils/Shared_Device_Ptr.hh"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ParameterList.hpp"

namespace cuda_mc
{

typedef cuda_profugus::Mesh_Geometry Geom;
typedef cuda_utils::Space_Vector     Space_Vector;

__global__ void test_tally_kernel( Cell_Tally<Geom> *tally,
                                   Geom             *geom,
                                   int               num_vals)
{
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if( tid < num_vals )
     {
         // Create particle
         Particle<Geom> p;
         p.set_group(0);
         p.set_matid(0);
         p.set_wt(1.0);
         p.live();

         // Create and initialize RNG state
         hiprandState_t rng_state;
         hiprand_init(tid,0,0,&rng_state);
         p.set_rng(&rng_state);
         
         // Initialize particle uniformly on [0, 1]
         double x_loc = hiprand_uniform_double(&rng_state);
         double y_loc = hiprand_uniform_double(&rng_state);
         double z_loc = hiprand_uniform_double(&rng_state);
         Space_Vector pos = {x_loc, y_loc, z_loc};

         // Direction doesn't matter
         Space_Vector dir = {1.0, 0.0, 0.0};

         geom->initialize(pos,dir,p.geo_state());

         // Tally with step length of 1.0
         tally->accumulate(1.0,p);

         // Move particle to new location
         pos.x = hiprand_uniform_double(&rng_state);
         pos.y = hiprand_uniform_double(&rng_state);
         pos.z = hiprand_uniform_double(&rng_state);

         geom->initialize(pos,dir,p.geo_state());

         // Tally with step length of 0.5
         tally->accumulate(0.5,p);
     }
}

void Cell_Tally_Tester::test_tally( const Vec_Dbl  &x_edges,
                                    const Vec_Dbl  &y_edges,
                                    const Vec_Dbl  &z_edges,
                                          RCP_XS     xs,
                                    const Vec_Int  &cells,
                                          Vec_Dbl  &tally_result,
                                          int       num_particles )
{
    // Build geometry
    auto geom = std::make_shared<Geom>(x_edges,
        y_edges,z_edges);
    std::vector<int> matids(geom->num_cells(),0);
    geom->set_matids(matids);
    cuda::Shared_Device_Ptr<Geom> sdp_geom(geom);

    REQUIRE( hipGetLastError() == hipSuccess );

    // Build physics
    Teuchos::RCP<Teuchos::ParameterList> pl( new Teuchos::ParameterList() );
    auto sdp_mat = cuda::shared_device_ptr<cuda_profugus::XS_Device>(*xs);

    REQUIRE( hipGetLastError() == hipSuccess );

    auto sp_phys = std::make_shared<Physics<Geom> >(pl,xs,sdp_mat);
    sp_phys->set_geometry(sdp_geom);
    cuda::Shared_Device_Ptr<Physics<Geom> > sdp_phys(sp_phys);

    REQUIRE( hipGetLastError() == hipSuccess );

    // Build cell tally
    std::cout << "Building Cell_Tally" << std::endl;
    auto sp_tally = std::make_shared<Cell_Tally<Geom> >(sdp_geom,sdp_phys);
    sp_tally->set_cells(cells);
    cuda::Shared_Device_Ptr<Cell_Tally<Geom> > tally(sp_tally);

    REQUIRE( hipGetLastError() == hipSuccess );

    // Launch kernel
    std::cout << "Launching kernel" << std::endl;
    int block_size = 256;
    int num_blocks = num_particles / block_size;
    if( num_blocks > 0 )
    {
        test_tally_kernel<<<num_blocks,block_size>>>( tally.get_device_ptr(),
                                                      sdp_geom.get_device_ptr(),
                                                      num_particles );
    }

    REQUIRE( hipGetLastError() == hipSuccess );
    hipDeviceSynchronize();

    if( num_particles % block_size != 0 )
    {
        test_tally_kernel<<<1,num_particles%block_size>>>( tally.get_device_ptr(),
                                                           sdp_geom.get_device_ptr(),
                                                           num_particles );
    }

    REQUIRE( hipGetLastError() == hipSuccess );
    hipDeviceSynchronize();

    std::cout << "Finalizing tallies" << std::endl;
    tally.update_host();
    sp_tally->finalize(num_particles);
    REQUIRE( hipGetLastError() == hipSuccess );

    // Copy tally result to host
    tally_result = sp_tally->results();
    REQUIRE( tally_result.size() == cells.size() );
}

} // end namespace cuda_mc

//---------------------------------------------------------------------------//
//                 end of Cell_Tally_Tester.cc
//---------------------------------------------------------------------------//
