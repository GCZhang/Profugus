#include "hip/hip_runtime.h"
//---------------------------------*-C++-*-----------------------------------//
/*!
 * \file   MC/cuda_geometry/test/Mesh_Geometry_Tester.cu
 * \author Steven Hamilton
 * \date   Mon Dec 14 13:28:26 2015
 * \brief  Mesh_Geometry_Tester class definitions.
 * \note   Copyright (c) 2015 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "gtest/Gtest_Functions.hh"
#include "utils/View_Field.hh"
#include "cuda_utils/Definitions.hh"
#include "cuda_utils/Device_Vector_device.t.hh"
#include "cuda_utils/CudaDBC.hh"

#include "../Mesh_Geometry.hh"
#include "Mesh_Geometry_Tester.hh"

typedef profugus::geometry::cell_type  cell_type;
typedef profugus::geometry::matid_type matid_type;
typedef cuda::Space_Vector             Point;
typedef cuda_profugus::Mesh_Geometry   Mesh_Geometry;

// Get volume from the mesh for each specified cell
__global__ void compute_volumes_kernel(Mesh_Geometry    mesh,
                                       int              num_points,
                                       const cell_type *cells,
                                       double          *volumes)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < num_points )
    {
        volumes[tid] = mesh.volume(cells[tid]);
    }
}

// Compute the matid for particle at each specified spatial location
__global__ void compute_matids_kernel(Mesh_Geometry   mesh,
                                      int             num_points,
                                      const Point    *points,
                                      matid_type     *matids)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < num_points )
    {
        // Create and initialize state on each thread
        // We're only testing matids so direction doesn't matter
        cuda_profugus::Mesh_State state;
        Point dir = {1.0, 0.0, 0.0};
        mesh.initialize(points[tid],dir,state);

        // Get matid
        matids[tid] = mesh.matid(state);
    }
}

namespace
{

// Build Mesh_Geometry
std::shared_ptr<Mesh_Geometry> get_mesh()
{
    std::vector<double> x_edges = {0.0, 0.1, 0.6, 0.9, 1.0};
    std::vector<double> y_edges = {-1.0, -0.6, 0.0};
    std::vector<double> z_edges = {2.0, 2.6, 3.4, 4.0};
    
    auto mesh = std::make_shared<Mesh_Geometry>(x_edges,y_edges,z_edges);
    return mesh;
}

}

//---------------------------------------------------------------------------//
// Compute volumes of specified cells
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_volume()
{
    auto mesh = get_mesh();

    std::vector<cell_type> host_cells = {4, 1, 22, 11};
    int num_points = host_cells.size();

    // Create memory on device
    typedef cuda::arch::Device Arch;
    cuda::Device_Vector<Arch,cell_type> device_cells(
        profugus::make_view(host_cells));
    cuda::Device_Vector<Arch,double> device_volumes(num_points);

    // Execute kernel
    compute_volumes_kernel<<<1,num_points>>>( *mesh,
                                               num_points,
                                               device_cells.data(),
                                               device_volumes.data());

    REQUIRE( hipGetLastError() == hipSuccess );

    // Copy volumes back to host
    std::vector<double> host_volumes(num_points);
    device_volumes.to_host(profugus::make_view(host_volumes));

    std::vector<double> expected_volumes = {0.1 * 0.6 * 0.6,
                                            0.5 * 0.4 * 0.6,
                                            0.3 * 0.6 * 0.6,
                                            0.1 * 0.4 * 0.8};

    EXPECT_VEC_SOFT_EQ( expected_volumes, host_volumes );
}

//---------------------------------------------------------------------------//
// Compute matids of specified cells
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_matid()
{
    auto mesh = get_mesh();

    std::vector<matid_type> all_matids = {1, 3, 2, 0,
                                          3, 1, 4, 1,
                                          2, 5, 2, 1,
                                          0, 1, 2, 3,
                                          1, 2, 3, 4,
                                          2, 3, 4, 5};

    std::vector<Point> host_points = {{0.7,  -0.9,  2.1},
                                      {0.5,  -0.5,  2.5},
                                      {0.99, -0.01, 3.99},
                                      {0.05, -0.8,  2.4}};

    int num_points = host_points.size();
        
    // Set matids with mesh (host call)
    mesh->set_matids(all_matids);

    // Create memroy on device
    typedef cuda::arch::Device Arch;
    cuda::Device_Vector<Arch,Point> device_points(
        profugus::make_view(host_points));
    cuda::Device_Vector<Arch,matid_type> device_cell_matids(num_points);

    // Execute kernel
    compute_matids_kernel<<<1,num_points>>>(
            *mesh,
             num_points,
             device_points.data(),
             device_cell_matids.data());

    REQUIRE( hipGetLastError() == hipSuccess );

    // Copy matids back to host
    std::vector<matid_type> host_cell_matids(num_points);
    device_cell_matids.to_host(profugus::make_view(host_cell_matids));

    std::vector<matid_type> expected_matids = {2, 1, 5, 1};

    EXPECT_VEC_EQ( expected_matids, host_cell_matids);
}

//---------------------------------------------------------------------------//
// end of MC/cuda_geometry/test/Mesh_Geometry_Tester.cu
//---------------------------------------------------------------------------//
